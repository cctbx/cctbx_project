#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : nanoBraggCUDA.cu
 Author      :
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <stdlib.h>
#include <stdio.h>
#include "nanotypes.h"
#include "cuda_compatibility.h"
#include <simtbx/nanoBragg/nanoBraggCUDA.cuh>
using simtbx::nanoBragg::shapetype;
using simtbx::nanoBragg::hklParams;
using simtbx::nanoBragg::SQUARE;
using simtbx::nanoBragg::ROUND;
using simtbx::nanoBragg::GAUSS;
using simtbx::nanoBragg::GAUSS_ARGCHK;
using simtbx::nanoBragg::GAUSS_STAR;
using simtbx::nanoBragg::TOPHAT;

static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

#define THREADS_PER_BLOCK_X 128
#define THREADS_PER_BLOCK_Y 1
#define THREADS_PER_BLOCK_TOTAL (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y)
#define VECTOR_SIZE 4

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err) {
	if (err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) << "(" << err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

static hipError_t cudaMemcpyVectorDoubleToDevice(CUDAREAL *dst, double *src, size_t vector_items) {
	CUDAREAL * temp = new CUDAREAL[vector_items];
	for (size_t i = 0; i < vector_items; i++) {
		temp[i] = src[i];
	}
	hipError_t ret = hipMemcpy(dst, temp, sizeof(*dst) * vector_items, hipMemcpyHostToDevice);
	delete temp;
	return ret;
}

/* make a unit vector pointing in same direction and report magnitude (both args can be same vector) */
double cpu_unitize(double *vector, double *new_unit_vector);
double cpu_unitize(double * vector, double * new_unit_vector) {

	double v1 = vector[1];
	double v2 = vector[2];
	double v3 = vector[3];

	double mag = sqrt(v1 * v1 + v2 * v2 + v3 * v3);

	if (mag != 0.0) {
		/* normalize it */
		new_unit_vector[0] = mag;
		new_unit_vector[1] = v1 / mag;
		new_unit_vector[2] = v2 / mag;
		new_unit_vector[3] = v3 / mag;
	} else {
		/* can't normalize, report zero vector */
		new_unit_vector[0] = 0.0;
		new_unit_vector[1] = 0.0;
		new_unit_vector[2] = 0.0;
		new_unit_vector[3] = 0.0;
	}
	return mag;
}


__global__ void nanoBraggSpotsInitCUDAKernel(int spixels, int fpixesl, float * floatimage, float * omega_reduction, float * max_I_x_reduction,
		float * max_I_y_reduction, bool * rangemap);

__global__ void nanoBraggSpotsCUDAKernel(int spixels, int fpixels, int roi_xmin, int roi_xmax, int roi_ymin, int roi_ymax, int oversample, int point_pixel,
CUDAREAL pixel_size, CUDAREAL subpixel_size, int steps, CUDAREAL detector_thickstep, int detector_thicksteps, CUDAREAL detector_thick, CUDAREAL detector_mu,
		const CUDAREAL * __restrict__ sdet_vector, const CUDAREAL * __restrict__ fdet_vector, const CUDAREAL * __restrict__ odet_vector,
		const CUDAREAL * __restrict__ pix0_vector, int curved_detector, CUDAREAL distance, CUDAREAL close_distance, const CUDAREAL * __restrict__ beam_vector,
		CUDAREAL Xbeam, CUDAREAL Ybeam, CUDAREAL dmin, CUDAREAL phi0, CUDAREAL phistep, int phisteps, const CUDAREAL * __restrict__ spindle_vector, int sources,
		const CUDAREAL * __restrict__ source_X, const CUDAREAL * __restrict__ source_Y, const CUDAREAL * __restrict__ source_Z,
		const CUDAREAL * __restrict__ source_I, const CUDAREAL * __restrict__ source_lambda, const CUDAREAL * __restrict__ a0, const CUDAREAL * __restrict__ b0,
		const CUDAREAL * __restrict c0, shapetype xtal_shape, CUDAREAL mosaic_spread, int mosaic_domains, const CUDAREAL * __restrict__ mosaic_umats,
		CUDAREAL Na, CUDAREAL Nb,
		CUDAREAL Nc, CUDAREAL V_cell,
		CUDAREAL water_size, CUDAREAL water_F, CUDAREAL water_MW, CUDAREAL r_e_sqr, CUDAREAL fluence, CUDAREAL Avogadro, CUDAREAL spot_scale, int integral_form, CUDAREAL default_F,
		int interpolate, const CUDAREAL * __restrict__ Fhkl, const hklParams * __restrict__ Fhklparams, int nopolar, const CUDAREAL * __restrict__ polar_vector, CUDAREAL polarization, CUDAREAL fudge,
		const int unsigned short * __restrict__ maskimage, float * floatimage /*out*/, float * omega_reduction/*out*/, float * max_I_x_reduction/*out*/,
		float * max_I_y_reduction /*out*/, bool * rangemap);


extern "C" void nanoBraggSpotsCUDA(int deviceId, int spixels, int fpixels, int roi_xmin, int roi_xmax, int roi_ymin, int roi_ymax, int oversample, int point_pixel,
                double pixel_size, double subpixel_size, int steps, double detector_thickstep, int detector_thicksteps, double detector_thick, double detector_mu,
                double sdet_vector[4], double fdet_vector[4], double odet_vector[4], double pix0_vector[4], int curved_detector, double distance, double close_distance,
                double beam_vector[4], double Xbeam, double Ybeam, double dmin, double phi0, double phistep, int phisteps, double spindle_vector[4], int sources,
                double *source_X, double *source_Y, double * source_Z, double * source_I, double * source_lambda, double a0[4], double b0[4], double c0[4],
                shapetype xtal_shape, double mosaic_spread, int mosaic_domains, double * mosaic_umats, double Na, double Nb, double Nc, double V_cell,
                double water_size, double water_F, double water_MW, double r_e_sqr, double fluence, double Avogadro, int integral_form, double default_F,
                int interpolate, double *** Fhkl, int h_min, int h_max, int h_range, int k_min, int k_max, int k_range, int l_min, int l_max, int l_range, int hkls,
                int nopolar, double polar_vector[4], double polarization, double fudge, int unsigned short * maskimage, float * floatimage /*out*/,
                double * omega_sum/*out*/, int * sumn /*out*/, double * sum /*out*/, double * sumsqr /*out*/, double * max_I/*out*/, double * max_I_x/*out*/,
                double * max_I_y /*out*/, double spot_scale) {

	int total_pixels = spixels * fpixels;

    hipSetDevice(deviceId);

	/*allocate and zero reductions */
	bool * rangemap = (bool*) calloc(total_pixels, sizeof(bool));
	float * omega_reduction = (float*) calloc(total_pixels, sizeof(float));
	float * max_I_x_reduction = (float*) calloc(total_pixels, sizeof(float));
	float * max_I_y_reduction = (float*) calloc(total_pixels, sizeof(float));

	/* clear memory (TODO: consider this being optional) */
	memset(floatimage, 0, sizeof(typeof(*floatimage)) * total_pixels);

	/*create transfer arguments to device space*/
	int cu_spixels = spixels, cu_fpixels = fpixels;
	int cu_roi_xmin = roi_xmin, cu_roi_xmax = roi_xmax, cu_roi_ymin = roi_ymin, cu_roi_ymax = roi_ymax;
	int cu_oversample = oversample;
	int cu_point_pixel = point_pixel;
	CUDAREAL cu_pixel_size = pixel_size, cu_subpixel_size = subpixel_size;
	int cu_steps = steps;
	CUDAREAL cu_detector_thickstep = detector_thickstep, cu_detector_thick = detector_thick, cu_detector_mu = detector_mu;
	int cu_detector_thicksteps = detector_thicksteps;
	int cu_curved_detector = curved_detector;

	CUDAREAL cu_distance = distance, cu_close_distance = close_distance;

	CUDAREAL cu_Xbeam = Xbeam, cu_Ybeam = Ybeam;
	CUDAREAL cu_dmin = dmin, cu_phi0 = phi0, cu_phistep = phistep;
	int cu_phisteps = phisteps;

	shapetype cu_xtal_shape = xtal_shape;

	int cu_sources = sources;

	CUDAREAL cu_mosaic_spread = mosaic_spread;
	int cu_mosaic_domains = mosaic_domains;

	CUDAREAL cu_Na = Na, cu_Nb = Nb, cu_Nc = Nc, cu_V_cell = V_cell, cu_water_size = water_size, cu_water_F = water_F, cu_water_MW = water_MW;
	CUDAREAL cu_r_e_sqr = r_e_sqr, cu_fluence = fluence, cu_Avogadro = Avogadro, cu_spot_scale = spot_scale;

	int cu_integral_form = integral_form;
	CUDAREAL cu_default_F = default_F;
	int cu_interpolate = interpolate;

//	int cu_h_min = h_min, cu_h_max = h_max, cu_h_range = h_range;
//	int cu_k_min = k_min, cu_k_max = k_max, cu_k_range = k_range;
//	int cu_l_min = l_min, cu_l_max = l_max, cu_l_range = l_range;
//	int cu_hkls = hkls;

	int cu_nopolar = nopolar;
	CUDAREAL cu_polarization = polarization, cu_fudge = fudge;

	hklParams FhklParams = { hkls, h_min, h_max, h_range, k_min, k_max, k_range, l_min, l_max, l_range };
	hklParams * cu_FhklParams;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_FhklParams, sizeof(*cu_FhklParams)));
	CUDA_CHECK_RETURN(hipMemcpy(cu_FhklParams, &FhklParams, sizeof(*cu_FhklParams), hipMemcpyHostToDevice));

	const int vector_length = 4;
	CUDAREAL * cu_sdet_vector;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_sdet_vector, sizeof(*cu_sdet_vector) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_sdet_vector, sdet_vector, vector_length));

	CUDAREAL * cu_fdet_vector;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_fdet_vector, sizeof(*cu_fdet_vector) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_fdet_vector, fdet_vector, vector_length));

	CUDAREAL * cu_odet_vector;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_odet_vector, sizeof(*cu_odet_vector) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_odet_vector, odet_vector, vector_length));

	CUDAREAL * cu_pix0_vector;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_pix0_vector, sizeof(*cu_pix0_vector) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_pix0_vector, pix0_vector, vector_length));

	CUDAREAL * cu_beam_vector;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_beam_vector, sizeof(*cu_beam_vector) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_beam_vector, beam_vector, vector_length));

	CUDAREAL * cu_spindle_vector;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_spindle_vector, sizeof(*cu_spindle_vector) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_spindle_vector, spindle_vector, vector_length));

	CUDAREAL * cu_a0;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_a0, sizeof(*cu_a0) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_a0, a0, vector_length));

	CUDAREAL * cu_b0;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_b0, sizeof(*cu_b0) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_b0, b0, vector_length));

	CUDAREAL * cu_c0;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_c0, sizeof(*cu_c0) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_c0, c0, vector_length));

	//	Unitize polar vector before sending it to the GPU. Optimization do it only once here rather than multiple time per pixel in the GPU.
	CUDAREAL * cu_polar_vector;
	double polar_vector_unitized[4];
	cpu_unitize(polar_vector, polar_vector_unitized);
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_polar_vector, sizeof(*cu_polar_vector) * vector_length));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_polar_vector, polar_vector_unitized, vector_length));

	CUDAREAL * cu_source_X = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_source_X, sizeof(*cu_source_X) * sources));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_source_X, source_X, sources));

	CUDAREAL * cu_source_Y = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_source_Y, sizeof(*cu_source_Y) * sources));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_source_Y, source_Y, sources));

	CUDAREAL * cu_source_Z = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_source_Z, sizeof(*cu_source_Z) * sources));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_source_Z, source_Z, sources));

	CUDAREAL * cu_source_I = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_source_I, sizeof(*cu_source_I) * sources));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_source_I, source_I, sources));

	CUDAREAL * cu_source_lambda = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_source_lambda, sizeof(*cu_source_lambda) * sources));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_source_lambda, source_lambda, sources));

	CUDAREAL * cu_mosaic_umats = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_mosaic_umats, sizeof(*cu_mosaic_umats) * mosaic_domains * 9));
	CUDA_CHECK_RETURN(cudaMemcpyVectorDoubleToDevice(cu_mosaic_umats, mosaic_umats, mosaic_domains * 9));

	float * cu_floatimage = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_floatimage, sizeof(*cu_floatimage) * total_pixels));
	CUDA_CHECK_RETURN(hipMemcpy(cu_floatimage, floatimage, sizeof(*cu_floatimage) * total_pixels, hipMemcpyHostToDevice));

	float * cu_omega_reduction = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_omega_reduction, sizeof(*cu_omega_reduction) * total_pixels));
	CUDA_CHECK_RETURN(hipMemcpy(cu_omega_reduction, omega_reduction, sizeof(*cu_omega_reduction) * total_pixels, hipMemcpyHostToDevice));

	float * cu_max_I_x_reduction = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * total_pixels));
	CUDA_CHECK_RETURN(hipMemcpy(cu_max_I_x_reduction, max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * total_pixels, hipMemcpyHostToDevice));

	float * cu_max_I_y_reduction = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * total_pixels));
	CUDA_CHECK_RETURN(hipMemcpy(cu_max_I_y_reduction, max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * total_pixels, hipMemcpyHostToDevice));

	bool * cu_rangemap = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_rangemap, sizeof(*cu_rangemap) * total_pixels));
	CUDA_CHECK_RETURN(hipMemcpy(cu_rangemap, rangemap, sizeof(*cu_rangemap) * total_pixels, hipMemcpyHostToDevice));

	int unsigned short * cu_maskimage = NULL;
	if (maskimage != NULL) {
		CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_maskimage, sizeof(*cu_maskimage) * total_pixels));
		CUDA_CHECK_RETURN(hipMemcpy(cu_maskimage, maskimage, sizeof(*cu_maskimage) * total_pixels, hipMemcpyHostToDevice));
	}

	int hklsize = h_range * k_range * l_range;
	CUDAREAL * FhklLinear = (CUDAREAL*) calloc(hklsize, sizeof(*FhklLinear));
	for (int h = 0; h < h_range; h++) {
		for (int k = 0; k < k_range; k++) {
//			memcpy(FhklLinear + (h * k_range * l_range + k * l_range), Fhkl[h][k], sizeof(*FhklLinear) * l_range);
			for (int l = 0; l < l_range; l++) {

				//	convert Fhkl double to CUDAREAL
				FhklLinear[h * k_range * l_range + k * l_range + l] = Fhkl[h][k][l];
			}
		}
	}

	CUDAREAL * cu_Fhkl = NULL;
	CUDA_CHECK_RETURN(hipMalloc((void ** )&cu_Fhkl, sizeof(*cu_Fhkl) * hklsize));
	CUDA_CHECK_RETURN(hipMemcpy(cu_Fhkl, FhklLinear, sizeof(*cu_Fhkl) * hklsize, hipMemcpyHostToDevice));
    free(FhklLinear);

	//int deviceId = 0;
	CUDA_CHECK_RETURN(hipGetDevice(&deviceId));
	hipDeviceProp_t deviceProps = { 0 };
	CUDA_CHECK_RETURN(hipGetDeviceProperties(&deviceProps, deviceId));
	int smCount = deviceProps.multiProcessorCount;

//	CUDA_CHECK_RETURN(hipFuncSetCacheConfig(reinterpret_cast<const void*>(nanoBraggSpotsCUDAKernel), hipFuncCachePreferShared));
//	CUDA_CHECK_RETURN(hipFuncSetCacheConfig(reinterpret_cast<const void*>(nanoBraggSpotsCUDAKernel), hipFuncCachePreferL1));

	dim3 threadsPerBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);
	//  dim3 numBlocks((spixels - 1) / threadsPerBlock.x + 1, (fpixels - 1) / threadsPerBlock.y + 1);
	dim3 numBlocks(smCount * 8, 1);

	//  initialize the device memory within a kernel.
	//	nanoBraggSpotsInitCUDAKernel<<<numBlocks, threadsPerBlock>>>(cu_spixels, cu_fpixels, cu_floatimage, cu_omega_reduction, cu_max_I_x_reduction, cu_max_I_y_reduction, cu_rangemap);
	//  CUDA_CHECK_RETURN(hipPeekAtLastError());
	//  CUDA_CHECK_RETURN(hipDeviceSynchronize());

	nanoBraggSpotsCUDAKernel<<<numBlocks, threadsPerBlock>>>(cu_spixels, cu_fpixels, cu_roi_xmin, cu_roi_xmax, cu_roi_ymin, cu_roi_ymax, cu_oversample,
			cu_point_pixel, cu_pixel_size, cu_subpixel_size, cu_steps, cu_detector_thickstep, cu_detector_thicksteps, cu_detector_thick, cu_detector_mu,
			cu_sdet_vector, cu_fdet_vector, cu_odet_vector, cu_pix0_vector, cu_curved_detector, cu_distance, cu_close_distance, cu_beam_vector,
			cu_Xbeam, cu_Ybeam, cu_dmin, cu_phi0, cu_phistep, cu_phisteps, cu_spindle_vector,
			cu_sources, cu_source_X, cu_source_Y, cu_source_Z, cu_source_I, cu_source_lambda, cu_a0, cu_b0, cu_c0, cu_xtal_shape,
			cu_mosaic_spread, cu_mosaic_domains, cu_mosaic_umats, cu_Na, cu_Nb, cu_Nc, cu_V_cell, cu_water_size, cu_water_F, cu_water_MW, cu_r_e_sqr, cu_fluence, 
			cu_Avogadro, cu_spot_scale, cu_integral_form, cu_default_F, cu_interpolate, cu_Fhkl, cu_FhklParams,
			cu_nopolar, cu_polar_vector, cu_polarization, cu_fudge, cu_maskimage,
			cu_floatimage /*out*/, cu_omega_reduction/*out*/, cu_max_I_x_reduction/*out*/, cu_max_I_y_reduction /*out*/, cu_rangemap /*out*/);

	CUDA_CHECK_RETURN(hipPeekAtLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());

	CUDA_CHECK_RETURN(hipMemcpy(floatimage, cu_floatimage, sizeof(*cu_floatimage) * total_pixels, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(omega_reduction, cu_omega_reduction, sizeof(*cu_omega_reduction) * total_pixels, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(max_I_x_reduction, cu_max_I_x_reduction, sizeof(*cu_max_I_x_reduction) * total_pixels, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(max_I_y_reduction, cu_max_I_y_reduction, sizeof(*cu_max_I_y_reduction) * total_pixels, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipMemcpy(rangemap, cu_rangemap, sizeof(*cu_rangemap) * total_pixels, hipMemcpyDeviceToHost));

	CUDA_CHECK_RETURN(hipFree(cu_sdet_vector));
	CUDA_CHECK_RETURN(hipFree(cu_fdet_vector));
	CUDA_CHECK_RETURN(hipFree(cu_odet_vector));
	CUDA_CHECK_RETURN(hipFree(cu_pix0_vector));
	CUDA_CHECK_RETURN(hipFree(cu_beam_vector));
	CUDA_CHECK_RETURN(hipFree(cu_spindle_vector));
	CUDA_CHECK_RETURN(hipFree(cu_polar_vector));
	CUDA_CHECK_RETURN(hipFree(cu_a0));
	CUDA_CHECK_RETURN(hipFree(cu_b0));
	CUDA_CHECK_RETURN(hipFree(cu_c0));
	CUDA_CHECK_RETURN(hipFree(cu_source_X));
	CUDA_CHECK_RETURN(hipFree(cu_source_Y));
	CUDA_CHECK_RETURN(hipFree(cu_source_Z));
	CUDA_CHECK_RETURN(hipFree(cu_source_I));
	CUDA_CHECK_RETURN(hipFree(cu_source_lambda));
	CUDA_CHECK_RETURN(hipFree(cu_FhklParams));
	CUDA_CHECK_RETURN(hipFree(cu_mosaic_umats));
	CUDA_CHECK_RETURN(hipFree(cu_floatimage));
	CUDA_CHECK_RETURN(hipFree(cu_omega_reduction));
	CUDA_CHECK_RETURN(hipFree(cu_max_I_x_reduction));
	CUDA_CHECK_RETURN(hipFree(cu_max_I_y_reduction));
	CUDA_CHECK_RETURN(hipFree(cu_maskimage));
	CUDA_CHECK_RETURN(hipFree(cu_rangemap));
    CUDA_CHECK_RETURN(hipFree(cu_Fhkl));

	*max_I = 0;
	*max_I_x = 0;
	*max_I_y = 0;
	*sum = 0.0;
	*sumsqr = 0.0;
	*sumn = 0;
	*omega_sum = 0.0;

	for (int i = 0; i < total_pixels; i++) {
		if (!rangemap[i]) {
			continue;
		}
		float pixel = floatimage[i];
		if (pixel > (double) *max_I) {
			*max_I = pixel;
			*max_I_x = max_I_x_reduction[i];
			*max_I_y = max_I_y_reduction[i];
		}
		*sum += pixel;
		*sumsqr += pixel * pixel;
		++(*sumn);
		*omega_sum += omega_reduction[i];
	}
	free(rangemap);
	free(omega_reduction);
	free(max_I_x_reduction);
	free(max_I_y_reduction);
}

/* cubic spline interpolation functions */
__device__ static void polin2(CUDAREAL *x1a, CUDAREAL *x2a, CUDAREAL ya[4][4], CUDAREAL x1, CUDAREAL x2, CUDAREAL *y);
__device__ static void polin3(CUDAREAL *x1a, CUDAREAL *x2a, CUDAREAL *x3a, CUDAREAL ya[4][4][4], CUDAREAL x1, CUDAREAL x2, CUDAREAL x3, CUDAREAL *y);
/* rotate a 3-vector about a unit vector axis */
__device__ static CUDAREAL *rotate_axis(const CUDAREAL * __restrict__ v, CUDAREAL *newv, const CUDAREAL * __restrict__ axis, const CUDAREAL phi);
/* scale the magnitude of a vector */
__device__ static CUDAREAL vector_scale(CUDAREAL *vector, CUDAREAL *new_vector, CUDAREAL scale);
/* Fourier transform of a truncated lattice */
__device__ static CUDAREAL sincg(CUDAREAL x, CUDAREAL N);
//__device__ static CUDAREAL sincgrad(CUDAREAL x, CUDAREAL N);
/* Fourier transform of a sphere */
__device__ static CUDAREAL sinc3(CUDAREAL x);

__device__ __inline__ static int flatten3dindex(int x, int y, int z, int x_range, int y_range, int z_range);

__device__ __inline__ CUDAREAL quickFcell_ldg(int hkls, int h_max, int h_min, int k_max, int k_min, int l_min, int l_max, int h0, int k0, int l0, int h_range, int k_range, int l_range, CUDAREAL defaultF, const CUDAREAL * __restrict__ Fhkl);

__global__ void nanoBraggSpotsInitCUDAKernel(int spixels, int fpixels, float * floatimage, float * omega_reduction, float * max_I_x_reduction,
		float * max_I_y_reduction, bool * rangemap) {

	const int total_pixels = spixels * fpixels;
	const int fstride = gridDim.x * blockDim.x;
	const int sstride = gridDim.y * blockDim.y;
	const int stride = fstride * sstride;

	for (int pixIdx = (blockDim.y * blockIdx.y + threadIdx.y) * fstride + blockDim.x * blockIdx.x + threadIdx.x; pixIdx < total_pixels; pixIdx += stride) {
		const int fpixel = pixIdx % fpixels;
		const int spixel = pixIdx / fpixels;

		/* position in pixel array */
		int j = spixel * fpixels + fpixel;

		if (j < total_pixels) {
			floatimage[j] = 0;
			omega_reduction[j] = 0;
			max_I_x_reduction[j] = 0;
			max_I_y_reduction[j] = 0;
			rangemap[j] = false;
		}
	}
}

__global__ void nanoBraggSpotsCUDAKernel(int spixels, int fpixels, int roi_xmin, int roi_xmax, int roi_ymin, int roi_ymax, int oversample, int point_pixel,
CUDAREAL pixel_size, CUDAREAL subpixel_size, int steps, CUDAREAL detector_thickstep, int detector_thicksteps, CUDAREAL detector_thick, CUDAREAL detector_mu,
		const CUDAREAL * __restrict__ sdet_vector, const CUDAREAL * __restrict__ fdet_vector, const CUDAREAL * __restrict__ odet_vector,
		const CUDAREAL * __restrict__ pix0_vector, int curved_detector, CUDAREAL distance, CUDAREAL close_distance, const CUDAREAL * __restrict__ beam_vector,
		CUDAREAL Xbeam, CUDAREAL Ybeam, CUDAREAL dmin, CUDAREAL phi0, CUDAREAL phistep, int phisteps, const CUDAREAL * __restrict__ spindle_vector, int sources,
		const CUDAREAL * __restrict__ source_X, const CUDAREAL * __restrict__ source_Y, const CUDAREAL * __restrict__ source_Z,
		const CUDAREAL * __restrict__ source_I, const CUDAREAL * __restrict__ source_lambda, const CUDAREAL * __restrict__ a0, const CUDAREAL * __restrict__ b0,
		const CUDAREAL * __restrict c0, shapetype xtal_shape, CUDAREAL mosaic_spread, int mosaic_domains, const CUDAREAL * __restrict__ mosaic_umats,
		CUDAREAL Na, CUDAREAL Nb, CUDAREAL Nc, CUDAREAL V_cell, CUDAREAL water_size, CUDAREAL water_F, CUDAREAL water_MW, CUDAREAL r_e_sqr, CUDAREAL fluence,
		CUDAREAL Avogadro, CUDAREAL spot_scale, int integral_form, CUDAREAL default_F, int interpolate, const CUDAREAL * __restrict__ Fhkl, const hklParams * __restrict__ FhklParams, int nopolar, const CUDAREAL * __restrict__ polar_vector,
		CUDAREAL polarization, CUDAREAL fudge, const int unsigned short * __restrict__ maskimage, float * floatimage /*out*/, float * omega_reduction/*out*/,
		float * max_I_x_reduction/*out*/, float * max_I_y_reduction /*out*/, bool * rangemap) {

	__shared__ CUDAREAL s_dmin;

	__shared__ bool s_nopolar;

	__shared__ int s_phisteps;
	__shared__ CUDAREAL s_phi0, s_phistep;
	__shared__ int s_mosaic_domains;
	__shared__ CUDAREAL s_mosaic_spread;
	__shared__ shapetype s_xtal_shape;

	__shared__ CUDAREAL s_Na, s_Nb, s_Nc;
	__shared__ bool s_interpolate;
	__shared__ int s_hkls, s_h_max, s_h_min, s_k_max, s_k_min, s_l_max, s_l_min, s_h_range, s_k_range, s_l_range;

	if (threadIdx.x == 0 && threadIdx.y == 0) {

		s_dmin = dmin;

		s_nopolar = nopolar;

		s_phisteps = phisteps;
		s_phi0 = phi0;
		s_phistep = phistep;

		s_mosaic_domains = mosaic_domains;
		s_mosaic_spread = mosaic_spread;

		s_xtal_shape = xtal_shape;
		s_Na = Na;
		s_Nb = Nb;
		s_Nc = Nc;

		s_interpolate = interpolate;

		s_hkls = FhklParams->hkls;
		s_h_max = FhklParams->h_max;
		s_h_min = FhklParams->h_min;
		s_k_max = FhklParams->k_max;
		s_k_min = FhklParams->k_min;
		s_l_max = FhklParams->l_max;
		s_l_min = FhklParams->l_min;
		s_h_range = FhklParams->h_range;
		s_k_range = FhklParams->k_range;
		s_l_range = FhklParams->l_range;

	}
	__syncthreads();

	const int total_pixels = spixels * fpixels;
	const int fstride = gridDim.x * blockDim.x;
	const int sstride = gridDim.y * blockDim.y;
	const int stride = fstride * sstride;
//	const int tidx = blockDim.x * threadIdx.y * +threadIdx.x;

//	__shared__ int sharedVectors[THREADS_PER_BLOCK_TOTAL + 1][1][9];
//	__shared__ CUDAREAL sharedVectors[THREADS_PER_BLOCK_TOTAL + 1][1][VECTOR_SIZE];
//	CUDAREAL * tmpVector1 = sharedVectors[tidx][0];
//	CUDAREAL * tmpVector2 = sharedVectors[tidx][1];

	/* add background from something amorphous */
	CUDAREAL F_bg = water_F;
	CUDAREAL I_bg = F_bg * F_bg * r_e_sqr * fluence * water_size * water_size * water_size * 1e6 * Avogadro / water_MW;

//	hklParams[0] = h_min;
//	hklParams[1] = h_max;
//	hklParams[2] = h_range;
//	hklParams[3] = k_min;
//	hklParams[4] = k_max;
//	hklParams[5] = k_range;
//	hklParams[6] = l_min;
//	hklParams[7] = l_max;
//	hklParams[8] = l_range;

	for (int pixIdx = (blockDim.y * blockIdx.y + threadIdx.y) * fstride + blockDim.x * blockIdx.x + threadIdx.x; pixIdx < total_pixels; pixIdx += stride) {
		const int fpixel = pixIdx % fpixels;
		const int spixel = pixIdx / fpixels;

		/* allow for just one part of detector to be rendered */
		if (fpixel < roi_xmin || fpixel > roi_xmax || spixel < roi_ymin || spixel > roi_ymax) { //ROI region of interest
			continue;
		}

		/* position in pixel array */
		const int j = pixIdx;

		/* allow for the use of a mask */
		if (maskimage != NULL) {
			/* skip any flagged pixels in the mask */
			if (maskimage[j] == 0) {
				continue;
			}
		}

		/* reset photon count for this pixel */
		CUDAREAL I = I_bg;
		CUDAREAL omega_sub_reduction = 0.0;
		CUDAREAL max_I_x_sub_reduction = 0.0;
		CUDAREAL max_I_y_sub_reduction = 0.0;
		CUDAREAL polar = 0.0;
		if (s_nopolar) {
			polar = 1.0;
		}

		/* add this now to avoid problems with skipping later */
		// move this to the bottom to avoid accessing global device memory. floatimage[j] = I_bg;
		/* loop over sub-pixels */
		int subS, subF;
		for (subS = 0; subS < oversample; ++subS) { // Y voxel
			for (subF = 0; subF < oversample; ++subF) { // X voxel
				/* absolute mm position on detector (relative to its origin) */
				CUDAREAL Fdet = subpixel_size * (fpixel * oversample + subF) + subpixel_size / 2.0; // X voxel
				CUDAREAL Sdet = subpixel_size * (spixel * oversample + subS) + subpixel_size / 2.0; // Y voxel
				//                  Fdet = pixel_size*fpixel;
				//                  Sdet = pixel_size*spixel;

				max_I_x_sub_reduction = Fdet;
				max_I_y_sub_reduction = Sdet;

				int thick_tic;
				for (thick_tic = 0; thick_tic < detector_thicksteps; ++thick_tic) {
					/* assume "distance" is to the front of the detector sensor layer */
					CUDAREAL Odet = thick_tic * detector_thickstep; // Z Orthagonal voxel.

					/* construct detector subpixel position in 3D space */
					//                      pixel_X = distance;
					//                      pixel_Y = Sdet-Ybeam;
					//                      pixel_Z = Fdet-Xbeam;
					//CUDAREAL * pixel_pos = tmpVector1;
					CUDAREAL pixel_pos[4];
					pixel_pos[1] = Fdet * __ldg(&fdet_vector[1]) + Sdet * __ldg(&sdet_vector[1]) + Odet * __ldg(&odet_vector[1]) + __ldg(&pix0_vector[1]); // X
					pixel_pos[2] = Fdet * __ldg(&fdet_vector[2]) + Sdet * __ldg(&sdet_vector[2]) + Odet * __ldg(&odet_vector[2]) + __ldg(&pix0_vector[2]); // X
					pixel_pos[3] = Fdet * __ldg(&fdet_vector[3]) + Sdet * __ldg(&sdet_vector[3]) + Odet * __ldg(&odet_vector[3]) + __ldg(&pix0_vector[3]); // X
//					pixel_pos[1] = Fdet * fdet_vector[1] + Sdet * sdet_vector[1] + Odet * odet_vector[1] + pix0_vector[1]; // X
//					pixel_pos[2] = Fdet * fdet_vector[2] + Sdet * sdet_vector[2] + Odet * odet_vector[2] + pix0_vector[2]; // Y
//					pixel_pos[3] = Fdet * fdet_vector[3] + Sdet * sdet_vector[3] + Odet * odet_vector[3] + pix0_vector[3]; // Z
					if (curved_detector) {
						/* construct detector pixel that is always "distance" from the sample */
						CUDAREAL dbvector[4];
						dbvector[1] = distance * beam_vector[1];
						dbvector[2] = distance * beam_vector[2];
						dbvector[3] = distance * beam_vector[3];
						/* treat detector pixel coordinates as radians */
						CUDAREAL newvector[] = { 0.0, 0.0, 0.0, 0.0 };
						rotate_axis(dbvector, newvector, sdet_vector, pixel_pos[2] / distance);
						rotate_axis(newvector, pixel_pos, fdet_vector, pixel_pos[3] / distance);
						//                          rotate(vector,pixel_pos,0,pixel_pos[3]/distance,pixel_pos[2]/distance);
					}
					/* construct the diffracted-beam unit vector to this sub-pixel */
					//CUDAREAL * diffracted = tmpVector2;
					CUDAREAL diffracted[4];
					CUDAREAL airpath = unitize(pixel_pos, diffracted);

					/* solid angle subtended by a pixel: (pix/airpath)^2*cos(2theta) */
					CUDAREAL omega_pixel = pixel_size * pixel_size / airpath / airpath * close_distance / airpath;
					/* option to turn off obliquity effect, inverse-square-law only */
					if (point_pixel) {
						omega_pixel = 1.0 / airpath / airpath;
					}

					/* now calculate detector thickness effects */
					CUDAREAL capture_fraction = 1.0;
					if (detector_thick > 0.0 && detector_mu> 0.0) {
						/* inverse of effective thickness increase */
						CUDAREAL parallax = dot_product_ldg(odet_vector, diffracted);
						capture_fraction = exp(-thick_tic * detector_thickstep / detector_mu / parallax)
								- exp(-(thick_tic + 1) * detector_thickstep / detector_mu / parallax);
					}

					/* loop over sources now */
					int source;
					for (source = 0; source < sources; ++source) {

						/* retrieve stuff from cache */
						//CUDAREAL * incident = tmpVector1;
						CUDAREAL incident[4];
						incident[1] = -__ldg(&source_X[source]);
						incident[2] = -__ldg(&source_Y[source]);
						incident[3] = -__ldg(&source_Z[source]);
						CUDAREAL lambda = __ldg(&source_lambda[source]);
						CUDAREAL source_fraction = __ldg(&source_I[source]);

						/* construct the incident beam unit vector while recovering source distance */
						// TODO[Giles]: Optimization! We can unitize the source vectors before passing them in.
						unitize(incident, incident);
//						CUDAREAL source_path = unitize(incident, incident);
//						CUDAREAL source_path = norm3d(incident[1], incident[2], incident[3]);

//						CUDAREAL * d = tmpVector2;
//						d[0] = diffracted[0];
//						d[1] = diffracted[1];
//						d[2] = diffracted[2];
//						d[3] = diffracted[3];

						/* construct the scattering vector for this pixel */
//						CUDAREAL * scattering = tmpVector1;
						CUDAREAL scattering[4];
						scattering[1] = (diffracted[1] - incident[1]) / lambda;
						scattering[2] = (diffracted[2] - incident[2]) / lambda;
						scattering[3] = (diffracted[3] - incident[3]) / lambda;
//						CUDAREAL scattering[] = { 0.0, (diffracted[1] - incident[1]) / lambda, (diffracted[2] - incident[2]) / lambda, (diffracted[3]
//								- incident[3]) / lambda };

						/* sin(theta)/lambda is half the scattering vector length */
//						magnitude(scattering);
//						CUDAREAL stol = 0.5 * scattering[0];
						CUDAREAL stol = 0.5 * norm3d(scattering[1], scattering[2], scattering[3]);

						/* rough cut to speed things up when we aren't using whole detector */
						if (s_dmin > 0.0 && stol > 0.0) {
							if (s_dmin > 0.5 / stol) {
								continue;
							}
						}

						/* polarization factor */
						if (!s_nopolar) {
							/* need to compute polarization factor */
							polar = polarization_factor(polarization, incident, diffracted, polar_vector);
						} else {
							polar = 1.0;
						}

						/* sweep over phi angles */
						for (int phi_tic = 0; phi_tic < s_phisteps; ++phi_tic) {
							CUDAREAL phi = s_phistep * phi_tic + s_phi0;

//							CUDAREAL ap[] = { 0.0, 0.0, 0.0, 0.0 };
//							CUDAREAL bp[] = { 0.0, 0.0, 0.0, 0.0 };
//							CUDAREAL cp[] = { 0.0, 0.0, 0.0, 0.0 };
							CUDAREAL ap[4];
							CUDAREAL bp[4];
							CUDAREAL cp[4];

							/* rotate about spindle if necessary */
							rotate_axis_ldg(a0, ap, spindle_vector, phi);
							rotate_axis_ldg(b0, bp, spindle_vector, phi);
							rotate_axis_ldg(c0, cp, spindle_vector, phi);

							/* enumerate mosaic domains */
							for (int mos_tic = 0; mos_tic < s_mosaic_domains; ++mos_tic) {
								/* apply mosaic rotation after phi rotation */
								CUDAREAL a[4];
								CUDAREAL b[4];
								CUDAREAL c[4];
								if (s_mosaic_spread > 0.0) {
									rotate_umat_ldg(ap, a, &mosaic_umats[mos_tic * 9]);
									rotate_umat_ldg(bp, b, &mosaic_umats[mos_tic * 9]);
									rotate_umat_ldg(cp, c, &mosaic_umats[mos_tic * 9]);
								} else {
									a[1] = ap[1];
									a[2] = ap[2];
									a[3] = ap[3];
									b[1] = bp[1];
									b[2] = bp[2];
									b[3] = bp[3];
									c[1] = cp[1];
									c[2] = cp[2];
									c[3] = cp[3];
								}
								//                                  printf("%d %f %f %f\n",mos_tic,mosaic_umats[mos_tic*9+0],mosaic_umats[mos_tic*9+1],mosaic_umats[mos_tic*9+2]);
								//                                  printf("%d %f %f %f\n",mos_tic,mosaic_umats[mos_tic*9+3],mosaic_umats[mos_tic*9+4],mosaic_umats[mos_tic*9+5]);
								//                                  printf("%d %f %f %f\n",mos_tic,mosaic_umats[mos_tic*9+6],mosaic_umats[mos_tic*9+7],mosaic_umats[mos_tic*9+8]);

								/* construct fractional Miller indicies */

//								CUDAREAL * scat_s = tmpVector2;
//								scat_s[0] = scattering[0];
//								scat_s[1] = scattering[1];
//								scat_s[2] = scattering[2];
//								scat_s[3] = scattering[3];
//
//								CUDAREAL h = dot_product(a, scat_s);
//								CUDAREAL k = dot_product(b, scat_s);
//								CUDAREAL l = dot_product(c, scat_s);
								CUDAREAL h = dot_product(a, scattering);
								CUDAREAL k = dot_product(b, scattering);
								CUDAREAL l = dot_product(c, scattering);

								/* round off to nearest whole index */
								int h0 = ceil(h - 0.5);
								int k0 = ceil(k - 0.5);
								int l0 = ceil(l - 0.5);

								/* structure factor of the lattice (paralelpiped crystal)
								 F_latt = sin(M_PI*Na*h)*sin(M_PI*Nb*k)*sin(M_PI*Nc*l)/sin(M_PI*h)/sin(M_PI*k)/sin(M_PI*l);
								 */
								CUDAREAL F_latt = 1.0; // Shape transform for the crystal.
								CUDAREAL hrad_sqr = 0.0;
								if (s_xtal_shape == SQUARE) {
									/* xtal is a paralelpiped */
									if (Na > 1) {
//										F_latt *= sincgrad(h, s_Na);
										F_latt *= sincg(M_PI * h, s_Na);
									}
									if (Nb > 1) {
//										F_latt *= sincgrad(k, s_Nb);
										F_latt *= sincg(M_PI * k, s_Nb);
									}
									if (Nc > 1) {
//										F_latt *= sincgrad(l, s_Nc);
										F_latt *= sincg(M_PI * l, s_Nc);
									}
								} else {
									/* handy radius in reciprocal space, squared */
									hrad_sqr = (h - h0) * (h - h0) * Na * Na + (k - k0) * (k - k0) * Nb * Nb + (l - l0) * (l - l0) * Nc * Nc;
								}
								if (s_xtal_shape == ROUND) {
									/* use sinc3 for elliptical xtal shape,
									 correcting for sqrt of volume ratio between cube and sphere */
									F_latt = Na * Nb * Nc * 0.723601254558268 * sinc3(M_PI * sqrt(hrad_sqr * fudge));
								}
								if (s_xtal_shape == GAUSS) {
									/* fudge the radius so that volume and FWHM are similar to square_xtal spots */
									F_latt = Na * Nb * Nc * exp(-(hrad_sqr / 0.63 * fudge));
								}
                                if (s_xtal_shape == GAUSS_ARGCHK) {
                                   /* fudge the radius so that volume and FWHM are similar to square_xtal spots */
                                   double my_arg = hrad_sqr / 0.63 * fudge;
                                   if (my_arg<35.){ F_latt = Na * Nb * Nc * exp(-(my_arg));
                                   } else { F_latt = 0.; } // warps coalesce when blocks of 32 pixels have no Bragg signal
                                }
                                if (s_xtal_shape == GAUSS_STAR){
                                   CUDAREAL a_cross_b[] = { 0.0, 0.0, 0.0, 0.0 };
                                   CUDAREAL b_cross_c[] = { 0.0, 0.0, 0.0, 0.0 };
                                   CUDAREAL c_cross_a[] = { 0.0, 0.0, 0.0, 0.0 };
                                   cross_product(a,b,a_cross_b);
                                   cross_product(b,c,b_cross_c);
                                   cross_product(c,a,c_cross_a);
                                   /* new reciprocal-space cell vectors */
                                   CUDAREAL a_star_tic[] = {0,0,0,0};
                                   CUDAREAL b_star_tic[] = {0,0,0,0};
                                   CUDAREAL c_star_tic[] = {0,0,0,0};
                                   vector_scale(b_cross_c,a_star_tic,1e20/V_cell);
                                   vector_scale(c_cross_a,b_star_tic,1e20/V_cell);
                                   vector_scale(a_cross_b,c_star_tic,1e20/V_cell);
                                   //}
                                   CUDAREAL dh=h-h0;
                                   CUDAREAL dk=k-k0;
                                   CUDAREAL dl=l-l0;
                                   CUDAREAL dx_star = dh*a_star_tic[1] + dk*b_star_tic[1] + dl*c_star_tic[1];
                                   CUDAREAL dy_star = dh*a_star_tic[2] + dk*b_star_tic[2] + dl*c_star_tic[2];
                                   CUDAREAL dz_star = dh*a_star_tic[3] + dk*b_star_tic[3] + dl*c_star_tic[3];
                                   CUDAREAL Nvol=Na*Nb*Nc;
                                   CUDAREAL xtal_size_sq = pow(Nvol*V_cell, CUDAREAL(2)/CUDAREAL(3));
                                   CUDAREAL rad_star_sqr = ( dx_star*dx_star + dy_star*dy_star + dz_star*dz_star )
                                                  *xtal_size_sq;
                                   F_latt = Nvol*exp(-( rad_star_sqr *1.9 * fudge ));
                                }
								if (s_xtal_shape == TOPHAT) {
									/* make a flat-top spot of same height and volume as square_xtal spots */
									F_latt = Na * Nb * Nc * (hrad_sqr * fudge < 0.3969);
								}
								/* no need to go further if result will be zero? */
								if (F_latt == 0.0 && water_size == 0.0)
									continue;

								/* find nearest point on Ewald sphere surface? */
								if (integral_form) {

									/* need to calculate reciprocal matrix */
									/* various cross products */
									CUDAREAL a_cross_b[] = { 0.0, 0.0, 0.0, 0.0 };
									CUDAREAL b_cross_c[] = { 0.0, 0.0, 0.0, 0.0 };
									CUDAREAL c_cross_a[] = { 0.0, 0.0, 0.0, 0.0 };
									cross_product(a, b, a_cross_b);
									cross_product(b, c, b_cross_c);
									cross_product(c, a, c_cross_a);

									/* new reciprocal-space cell vectors */
									CUDAREAL a_star[] = { 0.0, 0.0, 0.0, 0.0 };
									CUDAREAL b_star[] = { 0.0, 0.0, 0.0, 0.0 };
									CUDAREAL c_star[] = { 0.0, 0.0, 0.0, 0.0 };
									vector_scale(b_cross_c, a_star, 1e20 / V_cell);
									vector_scale(c_cross_a, b_star, 1e20 / V_cell);
									vector_scale(a_cross_b, c_star, 1e20 / V_cell);

									/* reciprocal-space coordinates of nearest relp */
									CUDAREAL relp[] = { 0.0, 0.0, 0.0, 0.0 };
									relp[1] = h0 * a_star[1] + k0 * b_star[1] + l0 * c_star[1];
									relp[2] = h0 * a_star[2] + k0 * b_star[2] + l0 * c_star[2];
									relp[3] = h0 * a_star[3] + k0 * b_star[3] + l0 * c_star[3];
									//                                      d_star = magnitude(relp)

									/* reciprocal-space coordinates of center of Ewald sphere */
									CUDAREAL Ewald0[] = { 0.0, 0.0, 0.0, 0.0 };
									Ewald0[1] = -incident[1] / lambda / 1e10;
									Ewald0[2] = -incident[2] / lambda / 1e10;
									Ewald0[3] = -incident[3] / lambda / 1e10;
									//                                      1/lambda = magnitude(Ewald0)

									/* distance from Ewald sphere in lambda=1 units */
									CUDAREAL dEwald0[] = { 0.0, 0.0, 0.0, 0.0 };
									dEwald0[1] = relp[1] - Ewald0[1];
									dEwald0[2] = relp[2] - Ewald0[2];
									dEwald0[3] = relp[3] - Ewald0[3];
									magnitude(dEwald0);
									CUDAREAL d_r = dEwald0[0] - 1.0;

									/* unit vector of diffracted ray through relp */
									CUDAREAL diffracted0[] = { 0.0, 0.0, 0.0, 0.0 };
									unitize(dEwald0, diffracted0);

									/* intersection with detector plane */
									CUDAREAL xd = dot_product_ldg(fdet_vector, diffracted0);
									CUDAREAL yd = dot_product_ldg(sdet_vector, diffracted0);
									CUDAREAL zd = dot_product_ldg(odet_vector, diffracted0);

									/* where does the central direct-beam hit */
									CUDAREAL xd0 = dot_product_ldg(fdet_vector, incident);
									CUDAREAL yd0 = dot_product_ldg(sdet_vector, incident);
									CUDAREAL zd0 = dot_product_ldg(odet_vector, incident);

									/* convert to mm coordinates */
									CUDAREAL Fdet0 = distance * (xd / zd) + Xbeam;
									CUDAREAL Sdet0 = distance * (yd / zd) + Ybeam;

									//printf("GOTHERE %g %g   %g %g\n",Fdet,Sdet,Fdet0,Sdet0);
									CUDAREAL test = exp(-((Fdet - Fdet0) * (Fdet - Fdet0) + (Sdet - Sdet0) * (Sdet - Sdet0) + d_r * d_r) / 1e-8);
								} // end of integral form

								/* structure factor of the unit cell */
								CUDAREAL F_cell = default_F;
								if (s_interpolate) {
									int h0_flr = floor(h);
									int k0_flr = floor(k);
									int l0_flr = floor(l);

									if (((h - s_h_min + 3) > s_h_range) || (h - 2 < s_h_min) || ((k - s_k_min + 3) > s_k_range) || (k - 2 < s_k_min)
											|| ((l - s_l_min + 3) > s_l_range) || (l - 2 < s_l_min)) {
										//											if (babble) {
										//												babble = 0;
										//												printf("WARNING: out of range for three point interpolation: h,k,l,h0,k0,l0: %g,%g,%g,%d,%d,%d \n", h, k, l, h0,
										//														k0, l0);
										//												printf("WARNING: further warnings will not be printed! ");
										//											}
										F_cell = quickFcell_ldg(s_hkls, s_h_max, s_h_min, s_k_max, s_k_min, s_l_max, s_l_min, h0, k0, l0, s_h_range, s_k_range, s_l_range, default_F, Fhkl);
									} else {
										/* integer versions of nearest HKL indicies */
										int h_interp[] = { 0, 0, 0, 0 };
										int k_interp[] = { 0, 0, 0, 0 };
										int l_interp[] = { 0, 0, 0, 0 };
										h_interp[0] = h0_flr - 1;
										h_interp[1] = h0_flr;
										h_interp[2] = h0_flr + 1;
										h_interp[3] = h0_flr + 2;
										k_interp[0] = k0_flr - 1;
										k_interp[1] = k0_flr;
										k_interp[2] = k0_flr + 1;
										k_interp[3] = k0_flr + 2;
										l_interp[0] = l0_flr - 1;
										l_interp[1] = l0_flr;
										l_interp[2] = l0_flr + 1;
										l_interp[3] = l0_flr + 2;

										/* polin function needs doubles */
										CUDAREAL h_interp_d[] = { 0.0, 0.0, 0.0, 0.0 };
										CUDAREAL k_interp_d[] = { 0.0, 0.0, 0.0, 0.0 };
										CUDAREAL l_interp_d[] = { 0.0, 0.0, 0.0, 0.0 };
										h_interp_d[0] = (CUDAREAL) h_interp[0];
										h_interp_d[1] = (CUDAREAL) h_interp[1];
										h_interp_d[2] = (CUDAREAL) h_interp[2];
										h_interp_d[3] = (CUDAREAL) h_interp[3];
										k_interp_d[0] = (CUDAREAL) k_interp[0];
										k_interp_d[1] = (CUDAREAL) k_interp[1];
										k_interp_d[2] = (CUDAREAL) k_interp[2];
										k_interp_d[3] = (CUDAREAL) k_interp[3];
										l_interp_d[0] = (CUDAREAL) l_interp[0];
										l_interp_d[1] = (CUDAREAL) l_interp[1];
										l_interp_d[2] = (CUDAREAL) l_interp[2];
										l_interp_d[3] = (CUDAREAL) l_interp[3];

										/* now populate the "y" values (nearest four structure factors in each direction) */
										CUDAREAL sub_Fhkl[4][4][4];
										int i1, i2, i3;
										for (i1 = 0; i1 < 4; i1++) {
											for (i2 = 0; i2 < 4; i2++) {
												for (i3 = 0; i3 < 4; i3++) {
													sub_Fhkl[i1][i2][i3] = __ldg(
															&Fhkl[flatten3dindex(h_interp[i1] - s_h_min, k_interp[i2] - s_k_min, l_interp[i3] - s_l_min, s_h_range,
																	s_k_range, s_l_range)]);
												}
											}
										}

										/* run the tricubic polynomial interpolation */
										polin3(h_interp_d, k_interp_d, l_interp_d, sub_Fhkl, h, k, l, &F_cell);
									}
								} else {
//								if (!interpolate) {
//									if (hkls && (h0 <= hklParams[1]) && (h0 >= hklParams[0]) && (k0 <= hklParams[4]) && (k0 >= hklParams[3]) && (l0 <= hklParams[7]) && (l0 >= hklParams[6])) {
//										/* just take nearest-neighbor */
//										F_cell = __ldg(&Fhkl[flatten3dindex(h0 - hklParams[0], k0 - hklParams[3], l0 - hklParams[6], hklParams[2], hklParams[5], hklParams[8])]);
//									} else {
//										F_cell = default_F;  // usually zero
//									}
//								}
									F_cell = quickFcell_ldg(s_hkls, s_h_max, s_h_min, s_k_max, s_k_min, s_l_max, s_l_min, h0, k0, l0, s_h_range, s_k_range, s_l_range, default_F, Fhkl);
//									if (s_hkls && (h0 <= s_h_max) && (h0 >= s_h_min) && (k0 <= s_k_max) && (k0 >= s_k_min) && (l0 <= s_l_max) && (l0 >= s_l_min)) {
//										/* just take nearest-neighbor */
//										F_cell = __ldg(&Fhkl[flatten3dindex(h0 - s_h_min, k0 - s_k_min, l0 - s_l_min, s_h_range, s_k_range, s_l_range)]);
////										F_cell = __ldg(&Fhkl[flatten3dindex(h0 - __ldg(&FhklParams->h_min), k0 - __ldg(&FhklParams->k_min), l0 - __ldg(&FhklParams->l_min), s_h_range, s_k_range, s_l_range)]);
////										F_cell = __ldg(&Fhkl[flatten3dindex(h0 - FhklParams->h_min, k0 - FhklParams->k_min, l0 - FhklParams->l_min, FhklParams->h_range, FhklParams->k_range, FhklParams->l_range)]);
//									}
								}

								/* now we have the structure factor for this pixel */

								/* convert amplitudes into intensity (photons per steradian) */
								I += F_cell * F_cell * F_latt * F_latt * source_fraction * capture_fraction * omega_pixel;
								omega_sub_reduction += omega_pixel;
							}
							/* end of mosaic loop */
						}
						/* end of phi loop */
					}
					/* end of source loop */
				}
				/* end of detector thickness loop */
			}
			/* end of sub-pixel y loop */
		}
		/* end of sub-pixel x loop */
		const double photons = I_bg + (r_e_sqr * spot_scale * fluence * polar * I) / steps;
		floatimage[j] = photons;
		omega_reduction[j] = omega_sub_reduction; // shared contention
		max_I_x_reduction[j] = max_I_x_sub_reduction;
		max_I_y_reduction[j] = max_I_y_sub_reduction;
		rangemap[j] = true;
	}
}

__device__ __inline__ CUDAREAL quickFcell_ldg(int hkls, int h_max, int h_min, int k_max, int k_min, int l_max, int l_min, int h0, int k0, int l0, int h_range, int k_range, int l_range, CUDAREAL defaultF, const CUDAREAL * __restrict__ Fhkl) {
	if (hkls && (h0 <= h_max) && (h0 >= h_min) && (k0 <= k_max) && (k0 >= k_min) && (l0 <= l_max) && (l0 >= l_min)) {
		/* just take nearest-neighbor */
//      F_cell = __ldg(&Fhkl[flatten3dindex(h0 - s_h_min, k0 - s_k_min, l0 - s_l_min, s_h_range, s_k_range, s_l_range)]);
		return __ldg(&Fhkl[flatten3dindex(h0 - h_min, k0 - k_min, l0 - l_min, h_range, k_range, l_range)]);
	} else {
		return defaultF;  // usually zero
	}
}

__device__ __inline__ int flatten3dindex(int x, int y, int z, int x_range, int y_range, int z_range) {
	return x * y_range * z_range + y * z_range + z;
}

/* rotate a point about a unit vector axis */
__device__ CUDAREAL *rotate_axis(const CUDAREAL * __restrict__ v, CUDAREAL * newv, const CUDAREAL * __restrict__ axis, const CUDAREAL phi) {

	const CUDAREAL sinphi = sin(phi);
	const CUDAREAL cosphi = cos(phi);
	const CUDAREAL a1 = axis[1];
	const CUDAREAL a2 = axis[2];
	const CUDAREAL a3 = axis[3];
	const CUDAREAL v1 = v[1];
	const CUDAREAL v2 = v[2];
	const CUDAREAL v3 = v[3];
	const CUDAREAL dot = (a1 * v1 + a2 * v2 + a3 * v3) * (1.0 - cosphi);

	newv[1] = a1 * dot + v1 * cosphi + (-a3 * v2 + a2 * v3) * sinphi;
	newv[2] = a2 * dot + v2 * cosphi + (+a3 * v1 - a1 * v3) * sinphi;
	newv[3] = a3 * dot + v3 * cosphi + (-a2 * v1 + a1 * v2) * sinphi;

	return newv;
}

/* scale magnitude of provided vector */
__device__ CUDAREAL vector_scale(CUDAREAL *vector, CUDAREAL *new_vector, CUDAREAL scale) {

	new_vector[1] = scale * vector[1];
	new_vector[2] = scale * vector[2];
	new_vector[3] = scale * vector[3];
	magnitude(new_vector);

	return new_vector[0];
}

/* Fourier transform of a grating */
__device__ CUDAREAL sincg(CUDAREAL x, CUDAREAL N) {
	if (x != 0.0)
		return sin(x * N) / sin(x);

	return N;

}

__device__ CUDAREAL sincgrad(CUDAREAL x, CUDAREAL N) {
	if (x != 0.0)
		return sinpi(x * N) / sinpi(x);

	return N;
}

/* Fourier transform of a sphere */
__device__ CUDAREAL sinc3(CUDAREAL x) {
	if (x != 0.0)
		return 3.0 * (sin(x) / x - cos(x)) / (x * x);

	return 1.0;

}

__device__ void polin2(CUDAREAL *x1a, CUDAREAL *x2a, CUDAREAL ya[4][4], CUDAREAL x1, CUDAREAL x2, CUDAREAL *y) {
	int j;
	CUDAREAL ymtmp[4];
	for (j = 1; j <= 4; j++) {
		polint(x2a, ya[j - 1], x2, &ymtmp[j - 1]);
	}
	polint(x1a, ymtmp, x1, y);
}

__device__ void polin3(CUDAREAL *x1a, CUDAREAL *x2a, CUDAREAL *x3a, CUDAREAL ya[4][4][4], CUDAREAL x1, CUDAREAL x2, CUDAREAL x3, CUDAREAL *y) {
	int j;
	CUDAREAL ymtmp[4];

	for (j = 1; j <= 4; j++) {
		polin2(x2a, x3a, &ya[j - 1][0], x2, x3, &ymtmp[j - 1]);
	}
	polint(x1a, ymtmp, x1, y);
}


