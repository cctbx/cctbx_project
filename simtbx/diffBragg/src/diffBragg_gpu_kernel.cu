#include "hip/hip_runtime.h"
#include "diffBraggCUDA.h"
#include <simtbx/diffBragg/src/diffuse_util.h>
#include <stdio.h>

/* Fourier transform of a grating */
__device__ CUDAREAL sincg(CUDAREAL x, CUDAREAL N) {
	if (x != 0.0)
		return sin(x * N) / sin(x);
	return N;
}

__global__
void gpu_sum_over_steps(
        int Npix_to_model, unsigned int* panels_fasts_slows,
        CUDAREAL* floatimage,
        CUDAREAL* wavelenimage,
        CUDAREAL* d_Umat_images, CUDAREAL* d2_Umat_images,
        CUDAREAL* d_Bmat_images, CUDAREAL* d2_Bmat_images,
        CUDAREAL* d_Ncells_images, CUDAREAL* d2_Ncells_images,
        CUDAREAL* d_fcell_images, CUDAREAL* d2_fcell_images,
        CUDAREAL* d_eta_images,
        CUDAREAL* d2_eta_images,
        CUDAREAL* d_lambda_images, CUDAREAL* d2_lambda_images,
        CUDAREAL* d_panel_rot_images, CUDAREAL* d2_panel_rot_images,
        CUDAREAL* d_panel_orig_images, CUDAREAL* d2_panel_orig_images,
        CUDAREAL* d_fp_fdp_images,
        const int Nsteps, int _printout_fpixel, int _printout_spixel, bool _printout, CUDAREAL _default_F,
        int oversample, bool _oversample_omega, CUDAREAL subpixel_size, CUDAREAL pixel_size,
        CUDAREAL detector_thickstep, CUDAREAL _detector_thick, const CUDAREAL* __restrict__ close_distances, CUDAREAL detector_attnlen,
        int detector_thicksteps, int sources, int phisteps, int mosaic_domains,
        bool use_lambda_coefficients, CUDAREAL lambda0, CUDAREAL lambda1,
        MAT3 eig_U, MAT3 eig_O, MAT3 eig_B, MAT3 RXYZ,
        VEC3* dF_vecs,
        VEC3* dS_vecs,
        const MAT3* __restrict__ UMATS_RXYZ,
        MAT3* UMATS_RXYZ_prime,
        MAT3* UMATS_RXYZ_dbl_prime,
        MAT3* RotMats,
        MAT3* dRotMats,
        MAT3* d2RotMats,
        MAT3* UMATS,
        MAT3* dB_mats,
        MAT3* dB2_mats,
        MAT3* Amatrices,
        const CUDAREAL* __restrict__ source_X, const CUDAREAL* __restrict__ source_Y,
        const CUDAREAL* __restrict__ source_Z, const CUDAREAL* __restrict__ source_lambda,
        const CUDAREAL* __restrict__ source_I,
        CUDAREAL kahn_factor,
        CUDAREAL Na, CUDAREAL Nb, CUDAREAL Nc,
        CUDAREAL Nd, CUDAREAL Ne, CUDAREAL Nf,
        CUDAREAL phi0, CUDAREAL phistep,
        VEC3 spindle_vec, VEC3 _polarization_axis,
        int h_range, int k_range, int l_range,
        int h_max, int h_min, int k_max, int k_min, int l_max, int l_min, CUDAREAL dmin,
        CUDAREAL fudge, bool complex_miller, int verbose, bool only_save_omega_kahn,
        bool isotropic_ncells, bool compute_curvatures,
        const CUDAREAL* __restrict__ _FhklLinear, const CUDAREAL* __restrict__ _Fhkl2Linear,
        bool* refine_Bmat, bool* refine_Ncells, bool refine_Ncells_def, bool* refine_panel_origin, bool* refine_panel_rot,
        bool refine_fcell, bool* refine_lambda, bool refine_eta, bool* refine_Umat,
        const CUDAREAL* __restrict__ fdet_vectors, const CUDAREAL* __restrict__ sdet_vectors,
        const CUDAREAL* __restrict__ odet_vectors, const CUDAREAL* __restrict__ pix0_vectors,
        bool _nopolar, bool _point_pixel, CUDAREAL _fluence, CUDAREAL _r_e_sqr, CUDAREAL _spot_scale, int Npanels,
        bool aniso_eta, bool no_Nabc_scale,
        const CUDAREAL* __restrict__ fpfdp,
        const CUDAREAL* __restrict__ fpfdp_derivs,
        const CUDAREAL* __restrict__ atom_data, int num_atoms, bool refine_fp_fdp,
        const int* __restrict__ nominal_hkl, bool use_nominal_hkl, MAT3 anisoU, MAT3 anisoG, MAT3 rotate_principal_axes,
        bool use_diffuse, CUDAREAL* d_diffuse_gamma_images, CUDAREAL* d_diffuse_sigma_images, bool refine_diffuse, bool gamma_miller_units,
        bool refine_Icell, bool save_wavelenimage, int laue_group_num, int stencil_size,
        bool Fhkl_gradient_mode, bool Fhkl_errors_mode, bool using_trusted_mask, bool Fhkl_channels_empty, bool Fhkl_have_scale_factors,
        int Num_ASU,
        const CUDAREAL* __restrict__ data_residual, const CUDAREAL* __restrict__ data_variance,
        const int* __restrict__ data_freq, const bool* __restrict__ data_trusted,
        const int* __restrict__ FhklLinear_ASUid,
        const CUDAREAL* __restrict__ Fhkl_channels,
        const CUDAREAL* __restrict__ Fhkl_scale, CUDAREAL* Fhkl_scale_deriv,
        bool gaussian_star_shape, bool square_shape)
{ // BEGIN GPU kernel

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_stride = blockDim.x * gridDim.x;
    __shared__ CUDAREAL s_phi0, s_phistep, gx,gy,gz;
    __shared__ int s_phisteps;
    __shared__ bool s_gaussian_star_shape;
    __shared__ bool s_square_shape;
    __shared__ bool s_Fhkl_channels_empty;
    __shared__ bool s_Fhkl_have_scale_factors;
    __shared__ bool s_Fhkl_gradient_mode;
    __shared__ bool s_Fhkl_errors_mode;
    __shared__ int s_Num_ASU;
    __shared__ bool s_refine_Icell;
    __shared__ bool s_use_diffuse;
    __shared__ bool s_use_nominal_hkl;
    __shared__ bool s_refine_fp_fdp;
    __shared__ bool s_complex_miller;
    __shared__ int s_num_atoms;
    __shared__ bool s_aniso_eta;
    __shared__ bool s_no_Nabc_scale;
    __shared__ bool s_compute_curvatures;
    __shared__ MAT3 s_Ot;
    __shared__ MAT3 Ainv;
    __shared__ bool s_refine_diffuse;
    __shared__ bool s_gamma_miller_units;
    __shared__ MAT3 _NABC;
    __shared__ MAT3 s_dN;
    __shared__ CUDAREAL C;
    __shared__ CUDAREAL two_C;
    __shared__ MAT3 Bmat_realspace;
    __shared__ MAT3 Amat_init;
    __shared__ CUDAREAL s_Na;
    __shared__ CUDAREAL s_Nb;
    __shared__ CUDAREAL s_Nc;
    __shared__ CUDAREAL s_NaNbNc_squared;
    __shared__ int s_h_max, s_k_max, s_l_max, s_h_min, s_k_min, s_l_min, s_h_range, s_k_range, s_l_range;
    __shared__ int s_oversample, s_detector_thicksteps, s_sources, s_mosaic_domains,  s_printout_fpixel,
        s_printout_spixel, s_verbose, s_Nsteps;
    __shared__ CUDAREAL s_detector_thickstep, s_detector_attnlen, s_subpixel_size, s_pixel_size, s_lambda0,
        s_lambda1, sX0, sY0, sZ0, s_detector_thick, s_default_F,  s_overall_scale, s_kahn_factor;
    __shared__ bool s_oversample_omega, s_printout, s_nopolar;
    __shared__ VEC3 s_polarization_axis;

    __shared__ bool s_refine_Umat[3];
    __shared__ bool s_refine_panel_origin[3];
    __shared__ bool s_refine_panel_rot[3];
    __shared__ bool s_refine_Ncells[3];
    __shared__ bool s_refine_eta;
    __shared__ bool s_refine_Ncells_def;
    __shared__ bool s_refine_fcell;
    __shared__ bool s_refine_Bmat[6];
    __shared__ bool s_refine_lambda[2];
    __shared__ double s_NABC_det, s_NABC_det_sq;
    __shared__ MAT3 anisoG_local;
    __shared__ MAT3 anisoU_local;
    __shared__ MAT3 laue_mats[24];
    __shared__ MAT3 dG_dgam[3];
    __shared__ int num_laue_mats;
    __shared__ int dhh, dkk, dll;
    __shared__ VEC3 Hmin, Hmax, dHH, Hrange;
    //extern __shared__ CUDAREAL det_vecs[];
    //__shared__ int det_stride;
    __shared__ CUDAREAL s_xtal_size_sq;

    if (threadIdx.x==0){ // TODO can we get speed gains by dividing up the following definitions over more threads ?
        s_phisteps = phisteps;
        s_phi0 = phi0;
        s_phistep = phistep;
        gx = spindle_vec[0];
        gy = spindle_vec[1];
        gz = spindle_vec[2];
        s_gaussian_star_shape = gaussian_star_shape;
        s_square_shape = square_shape;
        for (int i=0; i<3; i++){
            s_refine_Ncells[i] = refine_Ncells[i];
            s_refine_Umat[i] = refine_Umat[i];
            s_refine_panel_origin[i] = refine_panel_origin[i];
            s_refine_panel_rot[i] = refine_panel_rot[i];
        }
        s_Fhkl_channels_empty = Fhkl_channels_empty;
        s_Fhkl_have_scale_factors = Fhkl_have_scale_factors;
        s_Fhkl_gradient_mode = Fhkl_gradient_mode;
        s_Fhkl_errors_mode = Fhkl_errors_mode;
        s_Num_ASU = Num_ASU;
        s_refine_Icell = refine_Icell;
        s_use_nominal_hkl = use_nominal_hkl;
        s_aniso_eta = aniso_eta;
        s_no_Nabc_scale = no_Nabc_scale;
        s_complex_miller = complex_miller;
        s_refine_lambda[0] = refine_lambda[0];
        s_refine_lambda[1] = refine_lambda[1];
        for(int i=0; i<6; i++){
            s_refine_Bmat[i] = refine_Bmat[i];
        }
        s_use_diffuse = use_diffuse;
        s_num_atoms = num_atoms;
        s_refine_fcell = refine_fcell;
        s_refine_eta = refine_eta;
        s_refine_Ncells_def = refine_Ncells_def;
        s_compute_curvatures = compute_curvatures;
        s_refine_fp_fdp = refine_fp_fdp;
        s_refine_diffuse = refine_diffuse;
        s_gamma_miller_units = gamma_miller_units;

        Bmat_realspace = eig_B*1e10;
        s_Ot = eig_O.transpose();
        Amat_init = eig_U*Bmat_realspace*s_Ot;
        Ainv = eig_U*(Bmat_realspace.transpose().inverse())* (eig_O.inverse());
        _NABC << Na,Nd,Nf,
                Nd,Nb,Ne,
                Nf,Ne,Nc;
        s_NABC_det = _NABC.determinant(); // TODO is this slow ?
        s_NABC_det_sq = s_NABC_det*s_NABC_det;
        C = 2 / 0.63 * fudge;
        two_C = 2*C;
        s_Na = Na;
        s_Nb = Nb;
        s_Nc = Nc;
        s_NaNbNc_squared = (Na*Nb*Nc);
        s_NaNbNc_squared *= s_NaNbNc_squared;
        s_h_max = h_max;
        s_k_max = k_max;
        s_l_max = l_max;
        s_h_min = h_min;
        s_k_min = k_min;
        s_l_min = l_min;
        s_h_range = h_range;
        s_k_range = k_range;
        s_l_range = l_range;

        s_oversample = oversample;
        s_detector_thicksteps = detector_thicksteps;
        s_sources = sources;
        s_mosaic_domains = mosaic_domains;
        s_detector_thickstep = detector_thickstep;
        s_detector_attnlen = detector_attnlen;
        s_subpixel_size = subpixel_size;
        s_pixel_size = pixel_size;
        s_detector_thick = _detector_thick;
        s_lambda0 = lambda0;
        s_lambda1 = lambda1;
        s_oversample_omega = _oversample_omega;
        s_printout = _printout;
        s_printout_fpixel = _printout_fpixel;
        s_printout_spixel = _printout_spixel;
        s_pixel_size = pixel_size;
        s_default_F = _default_F;
        s_verbose = verbose;
        s_polarization_axis = _polarization_axis;
        s_kahn_factor = kahn_factor;
        s_nopolar = _nopolar;
        sX0 = source_X[0];
        sY0 = source_Y[0];
        sZ0 = source_Z[0];
        s_Nsteps = Nsteps;
        s_overall_scale = _r_e_sqr *_spot_scale * _fluence / Nsteps ;

        if (s_use_diffuse){
            anisoG_local = anisoG;
            anisoU_local = anisoU;

            num_laue_mats = gen_laue_mats(laue_group_num, laue_mats, rotate_principal_axes);
            for (int i_gam=0; i_gam<3; i_gam++){
              dG_dgam[i_gam] << 0,0,0,0,0,0,0,0,0;
              dG_dgam[i_gam](i_gam, i_gam) = 1;
            }
            if (s_gamma_miller_units){
              anisoG_local = anisoG_local * Bmat_realspace;
              for (int i_gam=0; i_gam<3; i_gam++){
                dG_dgam[i_gam] = dG_dgam[i_gam] * Bmat_realspace;
              }
            }
            dhh = dkk = dll = stencil_size; // Limits of stencil for diffuse calc
        }
        Hmin << s_h_min,s_k_min,s_l_min;
        Hmax << s_h_max,s_k_max,s_l_max;
        dHH << dhh,dkk,dll;
        Hrange << s_h_range,s_k_range,s_l_range;
        //det_stride = Npanels*3;
        //for(int i=0; i< det_stride; i++){
        //    det_vecs[i] = fdet_vectors[i];
        //    det_vecs[i+det_stride] = sdet_vectors[i];
        //}
        MAT3 Amat0 = Amatrices[0];
        VEC3 A(Amat0(0,0),Amat0(0,1), Amat0(0,2));
        VEC3 B(Amat0(1,0),Amat0(1,1), Amat0(1,2));
        VEC3 C(Amat0(2,0),Amat0(2,1), Amat0(2,2));
        CUDAREAL cell_vol = A.dot(B.cross(C));
        s_xtal_size_sq = pow(s_NABC_det*cell_vol, (CUDAREAL)2/(CUDAREAL)3);
    }

    //extern __shared__ CUDAREAL source_data[];
    //int threads_per_block = blockDim.x;
    //int num_source_blocks = (sources + threads_per_block-1)/threads_per_block;
    //int tx = threadIdx.x;
    //if (tx < threads_per_block){
    //    for (int i_source_block=0; i_source_block < num_source_blocks; i_source_block++){
    //        int idx = i_source_block*threads_per_block + tx;
    //        if (idx< sources){
    //            source_data[idx] = source_X[idx];
    //            source_data[sources+idx] = source_Y[idx];
    //            source_data[sources*2+idx] = source_Z[idx];
    //            source_data[sources*3+idx] = source_lambda[idx];
    //            source_data[sources*4+idx] = source_I[idx];
    //        }
    //    }
    //}

    __syncthreads();

    for (int i_pix=tid; i_pix < Npix_to_model; i_pix+= thread_stride){

        if (using_trusted_mask){
            if (!data_trusted[i_pix])
                continue;
        }

        int _pid = panels_fasts_slows[i_pix*3];
        int _fpixel = panels_fasts_slows[i_pix*3+1];
        int _spixel = panels_fasts_slows[i_pix*3+2];

        CUDAREAL Fhkl_deriv_coef=0;
        CUDAREAL Fhkl_hessian_coef=0;
        if (s_Fhkl_gradient_mode){
            CUDAREAL u = data_residual[i_pix];
            CUDAREAL one_by_v = 1/data_variance[i_pix];
            CUDAREAL Gterm = 1 - 2*u - u*u*one_by_v;
            Fhkl_deriv_coef = 0.5 * Gterm*one_by_v / data_freq[i_pix];
            if (s_Fhkl_errors_mode){
                Fhkl_hessian_coef = -0.5*one_by_v*(one_by_v*Gterm - 2  - 2*u*one_by_v -u*u*one_by_v*one_by_v)/data_freq[i_pix];
            }
        }

        //int fcell_idx=1;
        int nom_h, nom_k, nom_l;
        if (s_use_nominal_hkl){
            nom_h = nominal_hkl[i_pix*3];
            nom_k = nominal_hkl[i_pix*3+1];
            nom_l = nominal_hkl[i_pix*3+2];
        }
        CUDAREAL close_distance = close_distances[_pid];

        // reset photon count for this pixel
        double _I=0;
        double Ilambda=0;
        double Imiller_h=0;
        double Imiller_k=0;
        double Imiller_l=0;

        // reset derivative photon counts for the various parameters
        double rot_manager_dI[3] = {0,0,0};
        double rot_manager_dI2[3] = {0,0,0};
        double ucell_manager_dI[6]= {0,0,0,0,0,0};
        double ucell_manager_dI2[6]= {0,0,0,0,0,0};
        double Ncells_manager_dI[6]= {0,0,0,0,0,0};
        double Ncells_manager_dI2[6]= {0,0,0,0,0,0};
        double pan_orig_manager_dI[3]= {0,0,0};
        double pan_orig_manager_dI2[3]= {0,0,0};
        double pan_rot_manager_dI[3]= {0,0,0};
        double pan_rot_manager_dI2[3]= {0,0,0};
        double fcell_manager_dI = 0;
        double fcell_manager_dI2 = 0;
        double eta_manager_dI[3] = {0,0,0};
        double eta_manager_dI2[3] = {0,0,0};
        double lambda_manager_dI[2] = {0,0};
        double lambda_manager_dI2[2] = {0,0};
        double fp_fdp_manager_dI[2] = {0,0};
        double dI_diffuse[6] = {0,0,0,0,0,0};

        for(int _subS=0;_subS<s_oversample;++_subS){
        for(int _subF=0;_subF<s_oversample;++_subF){

            // absolute mm position on detector (relative to its origin)
            CUDAREAL _Fdet = s_subpixel_size*(_fpixel*s_oversample + _subF ) + s_subpixel_size/2.0;
            CUDAREAL _Sdet = s_subpixel_size*(_spixel*s_oversample + _subS ) + s_subpixel_size/2.0;

            // assume "distance" is to the front of the detector sensor layer
            int pid_x = _pid*3;
            int pid_y = _pid*3+1;
            int pid_z = _pid*3+2;

            CUDAREAL fx = fdet_vectors[pid_x];
            CUDAREAL fy = fdet_vectors[pid_y];
            CUDAREAL fz = fdet_vectors[pid_z];
            CUDAREAL sx = sdet_vectors[pid_x];
            CUDAREAL sy = sdet_vectors[pid_y];
            CUDAREAL sz = sdet_vectors[pid_z];
            CUDAREAL ox = odet_vectors[pid_x];
            CUDAREAL oy = odet_vectors[pid_y];
            CUDAREAL oz = odet_vectors[pid_z];
            CUDAREAL px = pix0_vectors[pid_x];
            CUDAREAL py = pix0_vectors[pid_y];
            CUDAREAL pz = pix0_vectors[pid_z];

            VEC3 _o_vec(ox, oy, oz);

        for(int _thick_tic=0;_thick_tic<s_detector_thicksteps;++_thick_tic){

            CUDAREAL _Odet = _thick_tic*s_detector_thickstep;

            CUDAREAL pixposX = _Fdet*fx + _Sdet*sx + _Odet*ox + px;
            CUDAREAL pixposY = _Fdet*fy + _Sdet*sy + _Odet*oy + py;
            CUDAREAL pixposZ = _Fdet*fz + _Sdet*sz + _Odet*oz + pz;
            VEC3 _pixel_pos(pixposX, pixposY, pixposZ);

            CUDAREAL _airpath = _pixel_pos.norm();
            VEC3 _diffracted = _pixel_pos/_airpath;

            // solid angle subtended by a pixel: (pix/airpath)^2*cos(2theta)
            CUDAREAL _omega_pixel = s_pixel_size*s_pixel_size/_airpath/_airpath*close_distance/_airpath;

            // option to turn off obliquity effect, inverse-square-law only
            if(_point_pixel) _omega_pixel = 1.0/_airpath/_airpath;

            // now calculate detector thickness effects
            CUDAREAL _capture_fraction = 1;

            if(s_detector_thick > 0.0 && s_detector_attnlen > 0.0)
            {
                // inverse of effective thickness increase
                CUDAREAL _parallax = _diffracted.dot(_o_vec) ; //dot_product(diffracted,odet_vector);
                _capture_fraction = exp(-_thick_tic*s_detector_thickstep/s_detector_attnlen/_parallax)
                                  -exp(-(_thick_tic+1)*s_detector_thickstep/s_detector_attnlen/_parallax);
            }
            CUDAREAL cap_frac_times_omega = _capture_fraction * _omega_pixel;

        for(int _source=0;_source<s_sources;++_source){
            //VEC3 _incident(-__ldg(&source_X[_source]),
            //               -__ldg(&source_Y[_source]),
            //               -__ldg(&source_Z[_source]));
            //VEC3 _incident(-source_data[_source],
            //               -source_data[s_sources+_source],
            //               -source_data[2*s_sources+_source]);
            //CUDAREAL _lambda = source_data[3*s_sources+_source];
            //CUDAREAL sI = source_data[4*s_sources+_source];
            VEC3 _incident(-source_X[_source],
                           -source_Y[_source],
                           -source_Z[_source]);
            CUDAREAL _lambda = source_lambda[_source];
            CUDAREAL sI = source_I[_source];
            //CUDAREAL _lambda = __ldg(&source_lambda[_source]);
            CUDAREAL lambda_ang = _lambda*1e10;
            if (use_lambda_coefficients){
                lambda_ang = s_lambda0 + s_lambda1*lambda_ang;
                _lambda = lambda_ang*1e-10;
            }

            // polarization
            CUDAREAL polar_for_Fhkl_grad=1;
            if (!s_nopolar && s_Fhkl_gradient_mode){
                //polar_for_Fhkl_grad = diffBragg_gpu_kernel_polarization(_incident, _diffracted,
                //                    s_polarization_axis, s_kahn_factor);
                // component of diffracted unit vector along incident beam unit vector
                CUDAREAL cos2theta = _incident.dot(_diffracted);
                CUDAREAL cos2theta_sqr = cos2theta*cos2theta;
                CUDAREAL sin2theta_sqr = 1-cos2theta_sqr;

                CUDAREAL _psi=0;
                if(s_kahn_factor != 0.0){
                    // cross product to get "vertical" axis that is orthogonal to the cannonical "polarization"
                    VEC3 B_in = s_polarization_axis.cross(_incident);
                    // cross product with incident beam to get E-vector direction
                    VEC3 E_in = _incident.cross(B_in);
                    // get components of diffracted ray projected onto the E-B plane
                    CUDAREAL _kEi = _diffracted.dot(E_in);
                    CUDAREAL _kBi = _diffracted.dot(B_in);
                    // compute the angle of the diffracted ray projected onto the incident E-B plane
                    _psi = -atan2(_kBi,_kEi);
                }
                // correction for polarized incident beam
                polar_for_Fhkl_grad = 0.5*(1.0 + cos2theta_sqr - s_kahn_factor*cos(2*_psi)*sin2theta_sqr);
            }

            VEC3 _scattering = (_diffracted - _incident) / _lambda;

            VEC3 q_vec(_scattering[0], _scattering[1], _scattering[2]);
            q_vec *= 1e-10;

            // TODO rename
            CUDAREAL texture_scale= 1;
            texture_scale *= cap_frac_times_omega;
            texture_scale *= sI;


        for (int _phi_tic=0; _phi_tic<s_phisteps; ++_phi_tic){
            MAT3 Rphi;
            CUDAREAL phi = s_phi0 + s_phistep*_phi_tic;
            if (phi != 0){
                CUDAREAL c = cos(phi);
                CUDAREAL omc = 1-c;
                CUDAREAL s = sin(phi);
                Rphi << c + gx*gx*omc,    gx*gy*omc-gz*s,   gx*gz*omc+gy*s,
                      gy*gx*omc + gz*s,   c + gy*gy*omc,   gy*gz*omc - gx*s,
                      gz*gx*omc - gy*s,  gz*gy*omc + gx*s, c + gz*gz*omc;
            }

        for(int _mos_tic=0;_mos_tic<s_mosaic_domains;++_mos_tic){
            int amat_idx = _mos_tic;
            MAT3 UBO = Amatrices[amat_idx];
            if (phi != 0){
                MAT3 Um = UMATS_RXYZ[_mos_tic]; // note, this will be slow - check if we can simply allow Um and Rphi to commute ...
                UBO = UBO*Um*Rphi.transpose()*Um.transpose();
            }

            VEC3 H_vec = UBO*q_vec;
            CUDAREAL _h = H_vec[0];
            CUDAREAL _k = H_vec[1];
            CUDAREAL _l = H_vec[2];

            int _h0 = ceil(_h - 0.5);
            int _k0 = ceil(_k - 0.5);
            int _l0 = ceil(_l - 0.5);

            VEC3 H0(_h0, _k0, _l0);

            VEC3 delta_H = H_vec - H0;
            VEC3 V = _NABC*delta_H;
            CUDAREAL _hrad_sqr = V.dot(V);
            CUDAREAL I0=0;
            if (s_square_shape){
                I0 = 1;
                if(s_Na>1)
                    I0 *= sincg(M_PI*_h,s_Na);
                if(s_Nb>1)
                    I0 *= sincg(M_PI*_k,s_Nb);
                if(s_Nc>1)
                    I0 *= sincg(M_PI*_l,s_Nc);
                I0 *= I0;
            }
            else{
                CUDAREAL exparg;
                if (s_gaussian_star_shape){
                    MAT3 Ainv = UBO.inverse();
                    VEC3 delta_Q = Ainv*delta_H;
                    CUDAREAL rad_star_sqr = delta_Q.dot(delta_Q)*s_xtal_size_sq;
                    exparg = rad_star_sqr*1.9*fudge;
                }
                else
                    exparg = _hrad_sqr*C/2;
                if (exparg< 35)
                    if (s_no_Nabc_scale)
                        I0 = exp(-2*exparg);
                    else
                        I0 = (s_NABC_det_sq)*exp(-2*exparg);
            }

            // are we doing diffuse scattering
            CUDAREAL step_diffuse_param[6]  = {0,0,0,0,0,0};
            if (s_use_diffuse){
              calc_diffuse_at_hkl(H_vec,H0,dHH,Hmin,Hmax,Hrange,Ainv,&_FhklLinear[0],num_laue_mats,laue_mats,anisoG_local,anisoU_local,dG_dgam,s_refine_diffuse,&I0,step_diffuse_param);
            } // end s_use_diffuse outer

            CUDAREAL _F_cell = s_default_F;
            CUDAREAL _F_cell2 = 0;
            int i_hklasu=0;

            if ( (_h0<=s_h_max) && (_h0>=s_h_min) && (_k0<=s_k_max) && (_k0>=s_k_min) && (_l0<=s_l_max) && (_l0>=s_l_min)  ) {
                int Fhkl_linear_index = (_h0-s_h_min) * s_k_range * s_l_range + (_k0-s_k_min) * s_l_range + (_l0-s_l_min);
                //_F_cell = __ldg(&_FhklLinear[Fhkl_linear_index]);
                _F_cell = _FhklLinear[Fhkl_linear_index];
                //if (complex_miller) _F_cell2 = __ldg(&_Fhkl2Linear[Fhkl_linear_index]);
                if (s_complex_miller) _F_cell2 = _Fhkl2Linear[Fhkl_linear_index];
                if (s_Fhkl_have_scale_factors) i_hklasu = FhklLinear_ASUid[Fhkl_linear_index];
            }


            CUDAREAL c_deriv_Fcell = 0;
            CUDAREAL d_deriv_Fcell = 0;
            if (s_complex_miller){
                CUDAREAL c_deriv_Fcell_real = 0;
                CUDAREAL c_deriv_Fcell_imag = 0;
                CUDAREAL d_deriv_Fcell_real = 0;
                CUDAREAL d_deriv_Fcell_imag = 0;
                if(s_num_atoms > 0){
                   CUDAREAL S_2 = 1.e-20*(_scattering[0]*_scattering[0]+_scattering[1]*_scattering[1]+_scattering[2]*_scattering[2]);

                    // fp is always followed by the fdp value
                   CUDAREAL val_fp = fpfdp[2*_source];
                   CUDAREAL val_fdp = fpfdp[2*_source+1];

                   CUDAREAL c_deriv_prime=0;
                   CUDAREAL c_deriv_dblprime=0;
                   CUDAREAL d_deriv_prime = 0;
                   CUDAREAL d_deriv_dblprime = 0;
                   if (s_refine_fp_fdp){
                   //   currently only supports two parameter model
                       int d_idx = 2*_source;
                       c_deriv_prime = fpfdp_derivs[d_idx];
                       c_deriv_dblprime = fpfdp_derivs[d_idx+1];
                       d_deriv_prime = fpfdp_derivs[d_idx+2*s_sources];
                       d_deriv_dblprime = fpfdp_derivs[d_idx+1+2*s_sources];
                   }

                   for (int  i_atom=0; i_atom < s_num_atoms; i_atom++){
                        // fractional atomic coordinates
                       CUDAREAL atom_x = atom_data[i_atom*5];
                       CUDAREAL atom_y = atom_data[i_atom*5+1];
                       CUDAREAL atom_z = atom_data[i_atom*5+2];
                       CUDAREAL B = atom_data[i_atom*5+3]; // B factor
                       B = exp(-B*S_2/4.0); // TODO: speed me up?
                       CUDAREAL occ = atom_data[i_atom*5+4]; // occupancy
                       CUDAREAL r_dot_h = _h0*atom_x + _k0*atom_y + _l0*atom_z;
                       CUDAREAL phase = 2*M_PI*r_dot_h;
                       CUDAREAL s_rdoth = sin(phase);
                       CUDAREAL c_rdoth = cos(phase);
                       CUDAREAL Bocc = B*occ;
                       CUDAREAL BC = B*c_rdoth;
                       CUDAREAL BS = B*s_rdoth;
                       CUDAREAL real_part = BC*val_fp - BS*val_fdp;
                       CUDAREAL imag_part = BS*val_fp + BC*val_fdp;
                       _F_cell += real_part;
                       _F_cell2 += imag_part;
                       if (s_refine_fp_fdp){
                            c_deriv_Fcell_real += BC*c_deriv_prime - BS*c_deriv_dblprime;
                            c_deriv_Fcell_imag += BS*c_deriv_prime + BC*c_deriv_dblprime;

                            d_deriv_Fcell_real += BC*d_deriv_prime - BS*d_deriv_dblprime;
                            d_deriv_Fcell_imag += BS*d_deriv_prime + BC*d_deriv_dblprime;
                       }
                   }
               }
               CUDAREAL Freal = _F_cell;
               CUDAREAL Fimag = _F_cell2;

               _F_cell = sqrt(Freal*Freal + Fimag*Fimag);
               if (s_refine_fp_fdp){
                   c_deriv_Fcell = Freal*c_deriv_Fcell_real + Fimag*c_deriv_Fcell_imag;
                   d_deriv_Fcell = Freal*d_deriv_Fcell_real + Fimag*d_deriv_Fcell_imag;
               }

            }
            if (!s_oversample_omega && ! s_Fhkl_gradient_mode)
                _omega_pixel = 1;

            CUDAREAL _I_cell = _F_cell;
            if (! s_refine_Icell)
                _I_cell *= _F_cell;
            CUDAREAL s_hkl=1;
            int Fhkl_channel=0;
            if (! s_Fhkl_channels_empty)
                Fhkl_channel = Fhkl_channels[_source];
            if (s_Fhkl_have_scale_factors)
                s_hkl = Fhkl_scale[i_hklasu + Fhkl_channel*s_Num_ASU];
            if (s_Fhkl_gradient_mode){
                CUDAREAL Fhkl_deriv_scale = s_overall_scale*polar_for_Fhkl_grad;
                CUDAREAL I_noFcell=texture_scale*I0;
                CUDAREAL dfhkl = I_noFcell*_I_cell * Fhkl_deriv_scale;
                CUDAREAL grad_incr = dfhkl*Fhkl_deriv_coef;
                int fhkl_grad_idx=i_hklasu + Fhkl_channel*s_Num_ASU;

                if (s_Fhkl_errors_mode){
                    // here we hi-kack the Fhkl_scale_deriv array, if computing errors, in order to store the hessian terms
                    // if we are getting the hessian terms, we no longer need the  gradients (e.g. by this point we are done refininig)
                    CUDAREAL hessian_incr = Fhkl_hessian_coef*dfhkl*dfhkl;
                    atomicAdd(&Fhkl_scale_deriv[fhkl_grad_idx], hessian_incr);
                }
                else{
                    atomicAdd(&Fhkl_scale_deriv[fhkl_grad_idx], grad_incr);
                }
                continue;
            }

            CUDAREAL _I_total = s_hkl*_I_cell *I0;
            CUDAREAL Iincrement = _I_total*texture_scale;
            _I += Iincrement;
            if (save_wavelenimage){
                Ilambda += Iincrement*lambda_ang;
                Imiller_h += Iincrement*_h;
                Imiller_k += Iincrement*_k;
                Imiller_l += Iincrement*_l;
            }

            if (s_refine_diffuse){
                CUDAREAL step_scale = texture_scale*_F_cell*_F_cell;
                for (int i_diff=0; i_diff <6; i_diff++){
                    dI_diffuse[i_diff] += step_scale*step_diffuse_param[i_diff];
                }
            }


            if (s_refine_fp_fdp){
                CUDAREAL I_noFcell = texture_scale*I0;
                fp_fdp_manager_dI[0] += 2*I_noFcell * (c_deriv_Fcell);
                fp_fdp_manager_dI[1] += 2*I_noFcell * (d_deriv_Fcell);
            }

            if(s_verbose > 3)
                printf("hkl= %f %f %f  hkl1= %d %d %d  Fcell=%f\n", _h,_k,_l,_h0,_k0,_l0, _F_cell);

            MAT3 UBOt;
            if (s_refine_Umat[0] || s_refine_Umat[1] ||s_refine_Umat[2] || s_refine_eta){
                UBOt = Amat_init;
                if (phi != 0)
                    UBOt = Rphi*UBOt;
            }
            if (s_refine_Umat[0]){
                MAT3 RyRzUBOt = RotMats[1]*RotMats[2]*UBOt;
                VEC3 delta_H_prime = (UMATS[_mos_tic]*dRotMats[0]*RyRzUBOt).transpose()*q_vec;
                CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
                CUDAREAL value = -two_C * V_dot_dV * Iincrement;
                CUDAREAL value2 =0;
                if (s_compute_curvatures) {
                    VEC3 delta_H_dbl_prime = (UMATS[_mos_tic]*d2RotMats[0]*RyRzUBOt).transpose()*q_vec;
                    CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                    CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                    value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
                }
                rot_manager_dI[0] += value;
                rot_manager_dI2[0] += value2;
            }
            if (s_refine_Umat[1]){
                MAT3 UmosRx = UMATS[_mos_tic]*RotMats[0];
                MAT3 RzUBOt = RotMats[2]*UBOt;
                VEC3 delta_H_prime =(UmosRx*dRotMats[1]*RzUBOt).transpose()*q_vec;
                CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
                CUDAREAL value = -two_C * V_dot_dV * Iincrement;

                CUDAREAL value2=0;
                if (s_compute_curvatures){
                    VEC3 delta_H_dbl_prime = (UmosRx*d2RotMats[1]*RzUBOt).transpose()*q_vec;
                    CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                    CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                    value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
                }
                rot_manager_dI[1] += value;
                rot_manager_dI2[1] += value2;
            }
            if (s_refine_Umat[2]){
                MAT3 UmosRxRy = UMATS[_mos_tic]*RotMats[0]*RotMats[1];
                VEC3 delta_H_prime = (UmosRxRy*dRotMats[2]*UBOt).transpose()*q_vec;
                CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
                CUDAREAL value = -two_C * V_dot_dV * Iincrement;

                CUDAREAL value2=0;
                if (s_compute_curvatures){
                    VEC3 delta_H_dbl_prime = (UmosRxRy*d2RotMats[2]*UBOt).transpose()*q_vec;
                    CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                    CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                    value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
                }
                rot_manager_dI[2] += value;
                rot_manager_dI2[2] += value2;
            }
            //Checkpoint for unit cell derivatives
            //MAT3 Ot = eig_O.transpose();
            MAT3 UmosRxRyRzU;
            VEC3 delta_H_prime;
            for(int i_uc=0; i_uc < 6; i_uc++ ){
                if (s_refine_Bmat[i_uc]){
                    UmosRxRyRzU = UMATS_RXYZ[_mos_tic]*eig_U;
                    delta_H_prime = ((UmosRxRyRzU*(dB_mats[i_uc])*s_Ot).transpose()*q_vec);
                    CUDAREAL V_dot_dV = V.dot(_NABC*delta_H_prime);
                    CUDAREAL value = -two_C * V_dot_dV * Iincrement;
                    CUDAREAL value2 =0;
                    if (s_compute_curvatures){
                        VEC3 delta_H_dbl_prime = ((UmosRxRyRzU*(dB2_mats[i_uc])*s_Ot).transpose()*q_vec);
                        CUDAREAL dV_dot_dV = (_NABC*delta_H_prime).dot(_NABC*delta_H_prime);
                        CUDAREAL dV2_dot_V = (_NABC*delta_H).dot(_NABC*delta_H_dbl_prime);
                        value2 = two_C*(two_C*V_dot_dV*V_dot_dV - dV2_dot_V - dV_dot_dV)*Iincrement;
                    }
                    ucell_manager_dI[i_uc] += value;
                    ucell_manager_dI2[i_uc] += value2;
                }
            } //end ucell deriv

            // Checkpoint for Ncells manager
            if (s_refine_Ncells[0]){
                int num_ncell_deriv = 1;
                if (! isotropic_ncells)
                    num_ncell_deriv = 3;
                for (int i_nc=0; i_nc < num_ncell_deriv; i_nc++) {
                    MAT3 dN;
                    dN << 0,0,0,0,0,0,0,0,0;
                    dN(i_nc, i_nc) = 1;
                    if (num_ncell_deriv == 1){
                        dN(0,0) = 1;
                        dN(1,1) = 1;
                        dN(2,2) = 1;
                    }
                    CUDAREAL N_i = _NABC(i_nc, i_nc);
                    VEC3 dV_dN = dN*delta_H;
                    CUDAREAL determ_deriv = (_NABC.inverse()*dN).trace(); // TODO speedops: precompute these, store shared var _NABC.inverse
                    CUDAREAL deriv_coef= determ_deriv - C* ( dV_dN.dot(V));
                    CUDAREAL value = 2*Iincrement*deriv_coef;
                    CUDAREAL value2=0;
                    if(s_compute_curvatures){
                        dN(i_nc, i_nc) = 0; // TODO check maths
                        value2 = ( -1/N_i/N_i - C*(dV_dN.dot(dV_dN))) *2*Iincrement;
                        value2 += deriv_coef*2*value;
                    }
                    Ncells_manager_dI[i_nc] += value;
                    Ncells_manager_dI2[i_nc] += value2;
                }
            } // end Ncells manager deriv

            if (s_refine_Ncells_def){
                for (int i_nc =3; i_nc < 6; i_nc++ ){
                    MAT3 dN;
                    if (i_nc ==3)
                        dN << 0,1,0,1,0,0,0,0,0;
                    else if (i_nc == 4)
                        dN << 0,0,0,0,0,1,0,1,0;
                    else
                        dN << 0,0,1,0,0,0,1,0,0;
                    VEC3 dV_dN = dN*delta_H;
                    CUDAREAL determ_deriv = (_NABC.inverse()*dN).trace(); // TODO speedops: precompute these
                    CUDAREAL deriv_coef = determ_deriv - C* (dV_dN.dot(V));
                    CUDAREAL value = 2*Iincrement*deriv_coef;
                    Ncells_manager_dI[i_nc] += value;
                    CUDAREAL value2 = 0;
                    if (s_compute_curvatures){
                        value2 = deriv_coef*value;
                        value2 +=  -2*C*Iincrement*(dV_dN.dot(dV_dN));
                    Ncells_manager_dI2[i_nc] += value2;
                    }
                }
            }

            // Checkpoint for Origin manager
            for (int i_pan_orig=0; i_pan_orig < 3; i_pan_orig++){
                if (s_refine_panel_origin[i_pan_orig]){
                    CUDAREAL per_k = 1/_airpath;
                    CUDAREAL per_k3 = pow(per_k,3);
                    CUDAREAL per_k5 = pow(per_k,5);
                    CUDAREAL lambda_ang = _lambda*1e10;

                    MAT3 M = -two_C*(_NABC*UBO)/lambda_ang;
                    VEC3 dk;
                    if (i_pan_orig == 0)
                        dk << 0,0,1;
                    else if (i_pan_orig == 1)
                        dk << 1,0,0;
                    else
                        dk << 0,1,0;

                    CUDAREAL G = dk.dot(_pixel_pos);
                    CUDAREAL pix2 = subpixel_size*subpixel_size;
                    VEC3 dk_hat = -per_k3*G*_pixel_pos + per_k*dk;
                    CUDAREAL coef = (M*dk_hat).dot(V);
                    CUDAREAL coef2 = -3*pix2*per_k5*G * (_o_vec.dot(_pixel_pos));
                    coef2 += pix2*per_k3*(_o_vec.dot(dk));
                    CUDAREAL value = coef*Iincrement + coef2*Iincrement/_omega_pixel;

                    pan_orig_manager_dI[i_pan_orig] += value;
                    pan_orig_manager_dI2[i_pan_orig] += 0;

                } // end origin manager deriv
            }

            for (int i_pan_rot=0; i_pan_rot < 3; i_pan_rot++){
                if(s_refine_panel_rot[i_pan_rot]){
                    CUDAREAL per_k = 1/_airpath;
                    CUDAREAL per_k3 = pow(per_k,3);
                    CUDAREAL per_k5 = pow(per_k,5);
                    CUDAREAL lambda_ang = _lambda*1e10;
                    MAT3 M = -two_C*(_NABC*UBO)/lambda_ang;
                    VEC3 dk = _Fdet*(dF_vecs[_pid*3 + i_pan_rot]) + _Sdet*(dS_vecs[_pid*3 + i_pan_rot]);
                    CUDAREAL G = dk.dot(_pixel_pos);
                    CUDAREAL pix2 = subpixel_size*subpixel_size;
                    VEC3 dk_hat = -per_k3*G*_pixel_pos + per_k*dk;
                    CUDAREAL coef = (M*dk_hat).dot(V);
                    CUDAREAL coef2 = -3*pix2*per_k5*G * (_o_vec.dot(_pixel_pos));
                    coef2 += pix2*per_k3*(_o_vec.dot(dk));
                    CUDAREAL value = coef*Iincrement + coef2*Iincrement/_omega_pixel;

                    pan_rot_manager_dI[i_pan_rot] += value;
                    pan_rot_manager_dI2[i_pan_rot] += 0;
                }
            }

            // checkpoint for Fcell manager
            if (s_refine_fcell){
                CUDAREAL value;
                if (s_refine_Icell)
                    value = I0* texture_scale;
                else
                    value = 2*I0*_F_cell * texture_scale; //Iincrement/_F_cell ;
                CUDAREAL value2=0;
                if (s_compute_curvatures){
                //    NOTE if _Fcell >0
                    value2 = 2*I0 * texture_scale;
                }
                //if (fcell_idx >=0 && fcell_idx <=2){
                if (s_use_nominal_hkl){
                    if (_h0==nom_h && _k0==nom_k && _l0==nom_l){
                        fcell_manager_dI += value;
                        fcell_manager_dI2 += value2;
                    }
                }
                else{
                    fcell_manager_dI += value;
                    fcell_manager_dI2 += value2;

                }
            } // end of fcell man deriv

            // checkpoint for eta manager
            if (s_refine_eta){
                for (int i_eta=0;i_eta<3; i_eta++){
                    if (i_eta > 0 && ! s_aniso_eta)
                        continue;
                    int mtic2 = _mos_tic + i_eta*s_mosaic_domains;
                    VEC3 DeltaH_deriv = (UMATS_RXYZ_prime[mtic2]*UBOt).transpose()*q_vec;
                    // vector V is _Nabc*Delta_H
                    VEC3 dV = _NABC*DeltaH_deriv;
                    CUDAREAL V_dot_dV = V.dot(dV);
                    CUDAREAL Iprime = -two_C*(V_dot_dV)*Iincrement;
                    eta_manager_dI[i_eta] += Iprime;
                    CUDAREAL Idbl_prime=0;
                    if (s_compute_curvatures){
                        VEC3 DeltaH_second_deriv = (UMATS_RXYZ_dbl_prime[mtic2]*UBOt).transpose()*q_vec;
                        VEC3 dV2 = _NABC*DeltaH_second_deriv;
                        Idbl_prime = -two_C*(dV.dot(dV) + V.dot(dV2))*Iincrement;
                        Idbl_prime += -two_C*(V_dot_dV)*Iprime;
                    }
                    eta_manager_dI2[i_eta] += Idbl_prime;
                }
            } // end of eta man deriv

            // checkpoint for lambda manager
            for(int i_lam=0; i_lam < 2; i_lam++){
                if (s_refine_lambda[i_lam]){
                    CUDAREAL lambda_ang = _lambda*1e10;
                    CUDAREAL NH_dot_V = (_NABC*H_vec).dot(V);
                    CUDAREAL dg_dlambda;
                    if (i_lam==0)
                        dg_dlambda = 1;
                    else // i_lam==1
                        dg_dlambda = lambda_ang;
                    CUDAREAL coef = NH_dot_V*two_C*(dg_dlambda) / lambda_ang;
                    CUDAREAL value = coef*Iincrement;
                    CUDAREAL value2 = 0;
                    lambda_manager_dI[i_lam] += value;
                    lambda_manager_dI2[i_lam] += value2;
                }
            }
            //end of lambda deriv
            if( s_printout){
             if( _subS==0 && _subF==0 && _thick_tic==0 && _source==0 &&  _mos_tic==0 ){
              if((_fpixel==s_printout_fpixel && _spixel==s_printout_spixel) || s_printout_fpixel < 0){
                   printf("%4d %4d :  lambda = %10.9g\n", _fpixel,_spixel, _lambda);
                   printf("at %g %g %g\n", _pixel_pos[0],_pixel_pos[1],_pixel_pos[2]);
                   printf("Fdet= %10.7g; Sdet= %10.7g ; Odet= %10.7g\n", _Fdet, _Sdet, _Odet);
                   printf("PIX0: %10.5g %10.5g %10.5g\n" , pix0_vectors[pid_x], pix0_vectors[pid_y], pix0_vectors[pid_z]);
                   printf("F: %10.5g %10.5g %10.5g\n" , fdet_vectors[pid_x], fdet_vectors[pid_y], fdet_vectors[pid_z]);
                   printf("S: %10.5g %10.5g %10.5g\n" , sdet_vectors[pid_x], sdet_vectors[pid_y], sdet_vectors[pid_z]);
                   printf("O: %10.5g %10.5g %10.5g\n" , odet_vectors[pid_x], odet_vectors[pid_y], odet_vectors[pid_z]);
                   printf("pid_x=%d, pid_y=%d; pid_z=%d\n", pid_x, pid_y, pid_z);

                   printf("QVECTOR: %10.5g %10.5g %10.5g\n" , q_vec[0], q_vec[1], q_vec[2]);
                   MAT3 UU = UMATS_RXYZ[_mos_tic];
                     printf("UMAT_RXYZ :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                      UU(0,0),  UU(0,1), UU(0,2),
                      UU(1,0),  UU(1,1), UU(1,2),
                      UU(2,0),  UU(2,1), UU(2,2));
                   UU = Bmat_realspace;
                     printf("Bmat_realspace :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                      UU(0,0),  UU(0,1), UU(0,2),
                      UU(1,0),  UU(1,1), UU(1,2),
                      UU(2,0),  UU(2,1), UU(2,2));
                   UU = UBO;
                     printf("UBO :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                      UU(0,0),  UU(0,1), UU(0,2),
                      UU(1,0),  UU(1,1), UU(1,2),
                      UU(2,0),  UU(2,1), UU(2,2));

                   UU = UBOt;
                     printf("UBOt :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                      UU(0,0),  UU(0,1), UU(0,2),
                      UU(1,0),  UU(1,1), UU(1,2),
                      UU(2,0),  UU(2,1), UU(2,2));

                   UU = UmosRxRyRzU;
                    printf("UmosRxRyRzU :\n%f  %f  %f\n%f  %f  %f\n%f  %f  %f\n",
                        UU(0,0),  UU(0,1), UU(0,2),
                        UU(1,0),  UU(1,1), UU(1,2),
                        UU(2,0),  UU(2,1), UU(2,2));
                   VEC3 AA = delta_H_prime;
                   printf("delta_H_prime :\n%f  %f  %f\n",
                        AA[0],  AA[1], AA[2]);
                   printf("Iincrement: %f\n", Iincrement);
                   printf("hkl= %f %f %f  hkl0= %d %d %d\n", _h,_k,_l,_h0,_k0,_l0);
                   printf(" F_cell=%g  F_cell2=%g I_latt=%g   I = %g\n", _F_cell,_F_cell2,I0,_I);
                   printf("I/steps %15.10g\n", _I/s_Nsteps);
                   //printf("Ilatt diffuse %15.10g\n", I_latt_diffuse);
                   printf("omega   %15.10g\n", _omega_pixel);
                   printf("default_F= %f\n", s_default_F);
                   printf("Incident[0]=%15.10g, Incident[1]=%15.10g, Incident[2]=%15.10g\n", _incident[0], _incident[1], _incident[2]);
                   printf("Diffracted[0]=%15.10g, Diffracted[1]=%15.10g, Diffracted[2]=%15.10g\n", _diffracted[0], _diffracted[1], _diffracted[2]);
                   printf("Scattering[0]=%15.10g, Scattering[1]=%15.10g, Scattering[2]=%15.10g\n", _scattering[0], _scattering[1], _scattering[2]);
                   printf("sourceI=%10.7g\n",  sI);
                  if (s_complex_miller)printf("COMPLEX MILLER!\n");
                  if (s_no_Nabc_scale)printf("No Nabc scale!\n");
                }
              }
            }

            } // end of mos_tic loop
            } // end of phi_tic loop
           } // end of source loop
          } // end of thick step loop
         } // end of fpos loop
        } // end of spos loop
        if (s_Fhkl_gradient_mode)
            continue;

        CUDAREAL _Fdet_ave = s_pixel_size*_fpixel + s_pixel_size/2.0;
        CUDAREAL _Sdet_ave = s_pixel_size*_spixel + s_pixel_size/2.0;
        CUDAREAL _Odet_ave = 0; //Odet; // TODO maybe make this more general for thick detectors?

        VEC3 _pixel_pos_ave(0,0,0);
        int pid_x = _pid*3;
        int pid_y = _pid*3+1;
        int pid_z = _pid*3+2;

        CUDAREAL fx = fdet_vectors[pid_x];
        CUDAREAL fy = fdet_vectors[pid_y];
        CUDAREAL fz = fdet_vectors[pid_z];

        CUDAREAL sx = sdet_vectors[pid_x];
        CUDAREAL sy = sdet_vectors[pid_y];
        CUDAREAL sz = sdet_vectors[pid_z];

        CUDAREAL ox = odet_vectors[pid_x];
        CUDAREAL oy = odet_vectors[pid_y];
        CUDAREAL oz = odet_vectors[pid_z];

        CUDAREAL px =pix0_vectors[pid_x];
        CUDAREAL py =pix0_vectors[pid_y];
        CUDAREAL pz =pix0_vectors[pid_z];

        _pixel_pos_ave[0] = _Fdet_ave * fx+_Sdet_ave*sx+_Odet_ave*ox+px;
        _pixel_pos_ave[1] = _Fdet_ave * fy+_Sdet_ave*sy+_Odet_ave*oy+py;
        _pixel_pos_ave[2] = _Fdet_ave * fz+_Sdet_ave*sz+_Odet_ave*oz+pz;

        CUDAREAL _airpath_ave = _pixel_pos_ave.norm();
        VEC3 _diffracted_ave = _pixel_pos_ave/_airpath_ave;
        CUDAREAL _omega_pixel_ave = s_pixel_size*s_pixel_size/_airpath_ave/_airpath_ave*close_distance/_airpath_ave;

        CUDAREAL _polar = 1;
        if (!s_nopolar){
            VEC3 _incident(-sX0, -sY0, -sZ0);
            _incident = _incident / _incident.norm();
            // component of diffracted unit vector along incident beam unit vector
            CUDAREAL cos2theta = _incident.dot(_diffracted_ave);
            CUDAREAL cos2theta_sqr = cos2theta*cos2theta;
            CUDAREAL sin2theta_sqr = 1-cos2theta_sqr;

            CUDAREAL _psi=0;
            if(kahn_factor != 0.0){
                // cross product to get "vertical" axis that is orthogonal to the cannonical "polarization"
                VEC3 B_in = s_polarization_axis.cross(_incident);
                // cross product with incident beam to get E-vector direction
                VEC3 E_in = _incident.cross(B_in);
                // get components of diffracted ray projected onto the E-B plane
                CUDAREAL _kEi = _diffracted_ave.dot(E_in);
                CUDAREAL _kBi = _diffracted_ave.dot(B_in);
                // compute the angle of the diffracted ray projected onto the incident E-B plane
                _psi = -atan2(_kBi,_kEi);
            }
            // correction for polarized incident beam
            _polar = 0.5*(1.0 + cos2theta_sqr - s_kahn_factor*cos(2*_psi)*sin2theta_sqr);
        }

        CUDAREAL _om = 1;
        if (!s_oversample_omega)
            _om=_omega_pixel_ave;
        // final scale term to being everything to photon number units
        CUDAREAL _scale_term = _polar*_om * s_overall_scale;
        floatimage[i_pix] = _scale_term*_I;
        if (save_wavelenimage){
            wavelenimage[i_pix*4] = Ilambda / _I;
            wavelenimage[i_pix*4+1] = Imiller_h / _I;
            wavelenimage[i_pix*4+2] = Imiller_k / _I;
            wavelenimage[i_pix*4+3] = Imiller_l / _I;
        }

        // udpate the rotation derivative images*
        for (int i_rot =0 ; i_rot < 3 ; i_rot++){
            if (s_refine_Umat[i_rot]){
                CUDAREAL value = _scale_term*rot_manager_dI[i_rot];
                CUDAREAL value2 = _scale_term*rot_manager_dI2[i_rot];
                int idx = i_rot*Npix_to_model + i_pix;
                d_Umat_images[idx] = value;
                d2_Umat_images[idx] = value2;
            }
        } // end rot deriv image increment

        //update the ucell derivative images
        for (int i_uc=0 ; i_uc < 6 ; i_uc++){
            if (s_refine_Bmat[i_uc]){
                CUDAREAL value = _scale_term*ucell_manager_dI[i_uc];
                CUDAREAL value2 = _scale_term*ucell_manager_dI2[i_uc];
                int idx= i_uc*Npix_to_model + i_pix;
                d_Bmat_images[idx] = value;
                d2_Bmat_images[idx] = value2;
            }
        }// end ucell deriv image increment

        //update the Ncells derivative image
        if (s_refine_Ncells[0]){
            CUDAREAL value = _scale_term*Ncells_manager_dI[0];
            CUDAREAL value2 = _scale_term*Ncells_manager_dI2[0];
            int idx = i_pix;
            d_Ncells_images[idx] = value;
            d2_Ncells_images[idx] = value2;

            if (! isotropic_ncells){
                value = _scale_term*Ncells_manager_dI[1];
                value2 = _scale_term*Ncells_manager_dI2[1];
                idx = Npix_to_model + i_pix;
                d_Ncells_images[idx] = value;
                d2_Ncells_images[idx] = value2;

                value = _scale_term*Ncells_manager_dI[2];
                value2 = _scale_term*Ncells_manager_dI2[2];
                idx = Npix_to_model*2 + i_pix;
                d_Ncells_images[idx] = value;
                d2_Ncells_images[idx] = value2;
            }
        }// end Ncells deriv image increment
        if (s_refine_Ncells_def){
            for (int i_nc=3; i_nc<6; i_nc++){
                CUDAREAL value = _scale_term*Ncells_manager_dI[i_nc];
                CUDAREAL value2 = _scale_term*Ncells_manager_dI2[i_nc];
                int idx = i_nc* Npix_to_model + i_pix;
                d_Ncells_images[idx] = value;
                d2_Ncells_images[idx] = value2;
            }
        }

        // update Fcell derivative image
        if(s_refine_fcell){
            CUDAREAL value = _scale_term*fcell_manager_dI;
            CUDAREAL value2 = _scale_term*fcell_manager_dI2;
            d_fcell_images[i_pix] = value;
            d2_fcell_images[i_pix] = value2;
        }// end Fcell deriv image increment

        if (s_refine_fp_fdp){
            // c derivative
            CUDAREAL value = _scale_term*fp_fdp_manager_dI[0];
            d_fp_fdp_images[i_pix] = value;
            // d derivative
            value = _scale_term*fp_fdp_manager_dI[1];
            d_fp_fdp_images[Npix_to_model + i_pix] = value;
        }
        if (s_refine_diffuse){
            for (int i_gam=0; i_gam < 3; i_gam++){
                CUDAREAL val = dI_diffuse[i_gam]*_scale_term;
                int img_idx = Npix_to_model*i_gam + i_pix;
                d_diffuse_gamma_images[img_idx] = val;
            }
            for (int i_sig=0; i_sig < 3; i_sig++){
                CUDAREAL val = dI_diffuse[i_sig+3]*_scale_term;
                int img_idx = Npix_to_model*i_sig + i_pix;
                d_diffuse_sigma_images[img_idx] = val;
            }
        }

        // update eta derivative image
        if(s_refine_eta){
            for (int i_eta=0; i_eta<3; i_eta++){
                if (i_eta > 0 && ! s_aniso_eta)
                    continue;
                int idx = i_pix + Npix_to_model*i_eta;
                CUDAREAL value = _scale_term*eta_manager_dI[i_eta];
                CUDAREAL value2 = _scale_term*eta_manager_dI2[i_eta];
                d_eta_images[idx] = value;
                d2_eta_images[idx] = value2;
            }
        }// end eta deriv image increment

        //update the lambda derivative images
        for (int i_lam=0 ; i_lam < 2 ; i_lam++){
            if (s_refine_lambda[i_lam]){
                CUDAREAL value = _scale_term*lambda_manager_dI[i_lam];
                CUDAREAL value2 = _scale_term*lambda_manager_dI2[i_lam];
                int idx = i_lam*Npix_to_model + i_pix;
                d_lambda_images[idx] = value;
                //d2_lambda_images[idx] = value2;
            }
        }// end lambda deriv image increment

        for (int i_pan_rot=0; i_pan_rot < 3; i_pan_rot++){
            if(s_refine_panel_rot[i_pan_rot]){
                CUDAREAL value = _scale_term*pan_rot_manager_dI[i_pan_rot];
                CUDAREAL value2 = _scale_term*pan_rot_manager_dI2[i_pan_rot];
                int idx = i_pan_rot*Npix_to_model + i_pix;
                d_panel_rot_images[idx] = value;
                //d2_panel_rot_images[idx] = value2;
            }
        }// end panel rot deriv image increment

        for (int i_pan_orig=0; i_pan_orig < 3; i_pan_orig++){
            if(s_refine_panel_origin[i_pan_orig]){
                CUDAREAL value = _scale_term*pan_orig_manager_dI[i_pan_orig];
                CUDAREAL value2 = _scale_term*pan_orig_manager_dI2[i_pan_orig];
                int idx = i_pan_orig*Npix_to_model + i_pix;
                d_panel_orig_images[idx] = value;
                //d2_panel_orig_images[idx] = value2;
            }
        }//end panel orig deriv image increment

    } // end i_pix loop

}  // END of GPU kernel
