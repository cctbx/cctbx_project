#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "diffBraggCUDA.h"
#include "diffBragg_gpu_kernel.h"
#include <stdio.h>
//lkalskdlaksdlkalsd

//#define BLOCKSIZE 128
//#define NUMBLOCKS 128
//https://stackoverflow.com/a/14038590/2077270
#define gpuErr(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void error_msg(hipError_t err, const char* msg){
    if (err != hipSuccess){
        printf("%s: CUDA error message: %s\n", msg, hipGetErrorString(err));
        exit(err);
    }
}

void diffBragg_sum_over_steps_cuda(
        int Npix_to_model,
        std::vector<unsigned int>& panels_fasts_slows,
        image_type& floatimage,
        images& d_image,
        images& d2_image,
        step_arrays& db_steps,
        detector& db_det,
        beam& db_beam,
        crystal& db_cryst,
        flags& db_flags,
        cuda_flags& db_cu_flags,
        diffBragg_cudaPointers& cp,
        timer_variables& TIMERS){

    int numblocks;
    int blocksize;
    char* diffBragg_blocks = getenv("DIFFBRAGG_NUM_BLOCKS");
    char* diffBragg_threads = getenv("DIFFBRAGG_THREADS_PER_BLOCK");
    if (diffBragg_threads==NULL)
        blocksize=128;
    else
        blocksize=atoi(diffBragg_threads);

    if (diffBragg_blocks==NULL)
        numblocks = (Npix_to_model+blocksize-1)/blocksize;
    else
        numblocks = atoi(diffBragg_blocks);

    int cuda_devices;
    hipGetDeviceCount(&cuda_devices);

    error_msg(hipGetLastError(), "after device count");
    if (db_flags.verbose > 1)
        printf("Found %d CUDA-capable devices\n", cuda_devices);

    //if (device_Id <= cuda_devices)
    gpuErr(hipSetDevice(db_cu_flags.device_Id));

    double time;
    struct timeval t1, t2;//, t3 ,t4;
    gettimeofday(&t1, 0);

//  determine if we need to allocate pixels, and how many.
//  For best usage, one should use the diffBragg property (visible from Python) Npix_to_allocate
//  in order to just allocate to the GPU - this is useful for ensemble refinement, where each shot
//  can have a variable number of pixels being modeled, and ony only needs to allocate the device once
//  (with the largest expected number of pixels for a given shot)
   // TODO clean up this logic a bit
    if (cp.device_is_allocated && (cp.npix_allocated < Npix_to_model)){
        printf("Need to re-allocate pixels, currently have %d allocated, but trying to model %d\n",
            cp.npix_allocated, Npix_to_model);
        exit(-1);
    }
    else if (db_cu_flags.Npix_to_allocate==-1){
        db_cu_flags.Npix_to_allocate = Npix_to_model;
    }
    else if (Npix_to_model > db_cu_flags.Npix_to_allocate){
        printf("Npix to model=%d is greater than the number of pixel requested for allocation (%d)!\n",
            Npix_to_model, db_cu_flags.Npix_to_allocate);
        exit(-1);
    }

//  support dynamic allocation for different numbers of sources
    if ( cp.previous_nsource != 0 && cp.previous_nsource != db_beam.number_of_sources){
        gpuErr(hipFree(cp.Fhkl_channels));
        gpuErr(hipFree(cp.cu_source_X));
        gpuErr(hipFree(cp.cu_source_Y));
        gpuErr(hipFree(cp.cu_source_Z));
        gpuErr(hipFree(cp.cu_source_I));
        gpuErr(hipFree(cp.cu_source_lambda));
        //printf("Reallocating for  %d sources!:\n", db_beam.number_of_sources);
        gpuErr(hipMallocManaged(&cp.Fhkl_channels, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_X, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_Y, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_Z, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_I, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_lambda, db_beam.number_of_sources*sizeof(CUDAREAL)));
        cp.previous_nsource = db_beam.number_of_sources;
    }

    if(cp.device_is_allocated){
        if (db_flags.verbose){
           printf("Will model %d pixels (GPU has %d pre-allocated pix)\n", Npix_to_model, cp.npix_allocated);
        }
    }
    else{
        if (db_flags.verbose){
           printf("Will model %d pixels and allocate %d pix\n", Npix_to_model, db_cu_flags.Npix_to_allocate);
        }
        // Check the Fhkl geradient arrays
        if (db_flags.Fhkl_have_scale_factors){
            gpuErr(hipMallocManaged(&cp.data_residual, db_cu_flags.Npix_to_allocate*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.data_variance, db_cu_flags.Npix_to_allocate*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.data_freq, db_cu_flags.Npix_to_allocate*sizeof(int)));
            gpuErr(hipMallocManaged(&cp.data_trusted, db_cu_flags.Npix_to_allocate*sizeof(bool)));
            gpuErr(hipMallocManaged(&cp.FhklLinear_ASUid, db_cryst.FhklLinear_ASUid.size()*sizeof(int)));
            gpuErr(hipMallocManaged(&cp.Fhkl_scale, d_image.Fhkl_scale.size()*sizeof(CUDAREAL)));
            // alloc Fhkl_scale_deriv to bs same length as Fhkl_scale.size(), as Fhkl_scale_deriv is only set when Fhkl_gradient_mode=True, typpically not first iteration
            gpuErr(hipMallocManaged(&cp.Fhkl_scale_deriv, d_image.Fhkl_scale.size()*sizeof(CUDAREAL)));
            cp.Fhkl_grad_arrays_allocated = true;
        }

        gpuErr(hipMallocManaged(&cp.Fhkl_channels, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_X, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_Y, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_Z, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_I, db_beam.number_of_sources*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_source_lambda, db_beam.number_of_sources*sizeof(CUDAREAL)));
        cp.previous_nsource = db_beam.number_of_sources;

        gpuErr(hipMallocManaged((void **)&cp.cu_UMATS, db_cryst.UMATS.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_UMATS_RXYZ, db_cryst.UMATS_RXYZ.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_AMATS, db_cryst.UMATS_RXYZ.size()*sizeof(MAT3)));
        if (db_cryst.UMATS_RXYZ_prime.size()>0)
            gpuErr(hipMallocManaged((void **)&cp.cu_UMATS_RXYZ_prime, db_cryst.UMATS_RXYZ_prime.size()*sizeof(MAT3)));
        if (db_cryst.UMATS_RXYZ_dbl_prime.size()>0)
            gpuErr(hipMallocManaged((void **)&cp.cu_UMATS_RXYZ_dbl_prime, db_cryst.UMATS_RXYZ_dbl_prime.size()*sizeof(MAT3)));

        gpuErr(hipMallocManaged((void **)&cp.cu_dB_Mats, db_cryst.dB_Mats.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_dB2_Mats, db_cryst.dB2_Mats.size()*sizeof(MAT3)));

        gpuErr(hipMallocManaged((void **)&cp.cu_RotMats, db_cryst.RotMats.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_dRotMats, db_cryst.dRotMats.size()*sizeof(MAT3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_d2RotMats, db_cryst.d2RotMats.size()*sizeof(MAT3)));

        gpuErr(hipMallocManaged(&cp.cu_fdet_vectors, db_det.fdet_vectors.size()*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_sdet_vectors, db_det.fdet_vectors.size()*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_odet_vectors, db_det.fdet_vectors.size()*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_pix0_vectors, db_det.fdet_vectors.size()*sizeof(CUDAREAL)));
        gpuErr(hipMallocManaged(&cp.cu_close_distances, db_det.close_distances.size()*sizeof(CUDAREAL)));

        if (db_cryst.fpfdp.size() > 0){
            gpuErr(hipMallocManaged(&cp.cu_fpfdp, db_cryst.fpfdp.size()*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.cu_atom_data, db_cryst.atom_data.size()*sizeof(CUDAREAL)));
        }
        if(db_cryst.fpfdp_derivs.size() > 0)
            gpuErr(hipMallocManaged(&cp.cu_fpfdp_derivs, db_cryst.fpfdp_derivs.size()*sizeof(CUDAREAL)));

        gpuErr(hipMallocManaged(&cp.cu_refine_Bmat, 6*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_Umat, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_Ncells, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_panel_origin, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_panel_rot, 3*sizeof(bool)));
        gpuErr(hipMallocManaged(&cp.cu_refine_lambda, 2*sizeof(bool)));

        gpuErr(hipMallocManaged(&cp.cu_Fhkl, db_cryst.FhklLinear.size()*sizeof(CUDAREAL)));
        if (db_flags.complex_miller)
            gpuErr(hipMallocManaged(&cp.cu_Fhkl2, db_cryst.FhklLinear.size()*sizeof(CUDAREAL)));

        gpuErr(hipMallocManaged((void **)&cp.cu_dF_vecs, db_det.dF_vecs.size()*sizeof(VEC3)));
        gpuErr(hipMallocManaged((void **)&cp.cu_dS_vecs, db_det.dF_vecs.size()*sizeof(VEC3)));

        //gettimeofday(&t3, 0));
        gpuErr(hipMallocManaged(&cp.cu_floatimage, db_cu_flags.Npix_to_allocate*sizeof(CUDAREAL) ));
        if (db_flags.wavelength_img){
            gpuErr(hipMallocManaged(&cp.cu_wavelenimage, 4*db_cu_flags.Npix_to_allocate*sizeof(CUDAREAL) ));
        }
        if (db_flags.refine_diffuse){
            gpuErr(hipMallocManaged(&cp.cu_d_diffuse_gamma_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.cu_d_diffuse_sigma_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL)));
        }
        if (db_flags.refine_fcell){
            gpuErr(hipMallocManaged(&cp.cu_d_fcell_images, db_cu_flags.Npix_to_allocate*1*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.cu_d2_fcell_images, db_cu_flags.Npix_to_allocate*1*sizeof(CUDAREAL)));
        }
        if (db_flags.refine_eta){
            gpuErr(hipMallocManaged(&cp.cu_d_eta_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.cu_d2_eta_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL)));
        }
        if (std::count(db_flags.refine_Umat.begin(), db_flags.refine_Umat.end(), true) > 0){
            gpuErr(hipMallocManaged(&cp.cu_d_Umat_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL) ));
            gpuErr(hipMallocManaged(&cp.cu_d2_Umat_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL) ));
        }
        if (std::count(db_flags.refine_Ncells.begin(), db_flags.refine_Ncells.end(), true) > 0 || db_flags.refine_Ncells_def){
            gpuErr(hipMallocManaged(&cp.cu_d_Ncells_images, db_cu_flags.Npix_to_allocate*6*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.cu_d2_Ncells_images, db_cu_flags.Npix_to_allocate*6*sizeof(CUDAREAL)));
        }
        if (std::count(db_flags.refine_panel_rot.begin(), db_flags.refine_panel_rot.end(), true) > 0)
            gpuErr(hipMallocManaged(&cp.cu_d_panel_rot_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL)));
        if (std::count(db_flags.refine_panel_origin.begin(), db_flags.refine_panel_origin.end(), true) > 0)
            gpuErr(hipMallocManaged(&cp.cu_d_panel_orig_images, db_cu_flags.Npix_to_allocate*3*sizeof(CUDAREAL)));
        if (std::count(db_flags.refine_lambda.begin(), db_flags.refine_lambda.end(), true) > 0)
            gpuErr(hipMallocManaged(&cp.cu_d_lambda_images, db_cu_flags.Npix_to_allocate*2*sizeof(CUDAREAL)));
        if (std::count(db_flags.refine_Bmat.begin(), db_flags.refine_Bmat.end(), true) > 0){
            gpuErr(hipMallocManaged(&cp.cu_d_Bmat_images, db_cu_flags.Npix_to_allocate*6*sizeof(CUDAREAL)));
            gpuErr(hipMallocManaged(&cp.cu_d2_Bmat_images, db_cu_flags.Npix_to_allocate*6*sizeof(CUDAREAL)));
        }
        if (db_flags.refine_fp_fdp)
            gpuErr(hipMallocManaged(&cp.cu_d_fp_fdp_images, db_cu_flags.Npix_to_allocate*2*sizeof(CUDAREAL)));
        if(db_cryst.nominal_hkl.size() >0)
            gpuErr(hipMallocManaged(&cp.cu_nominal_hkl, db_cu_flags.Npix_to_allocate*3*sizeof(int)));

        //gettimeofday(&t4, 0);
        //time = (1000000.0*(t4.tv_sec-t3.tv_sec) + t4.tv_usec-t3.tv_usec)/1000.0;
        //printf("TIME SPENT ALLOCATING (IMAGES ONLY):  %3.10f ms \n", time);
        gpuErr(hipMallocManaged(&cp.cu_panels_fasts_slows, db_cu_flags.Npix_to_allocate*3*sizeof(panels_fasts_slows[0])));
        cp.npix_allocated = db_cu_flags.Npix_to_allocate;
    } // END of allocation

    bool ALLOC = !cp.device_is_allocated; // shortcut variable

    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if (TIMERS.recording) TIMERS.cuda_alloc+= time;
    if(db_flags.verbose>1)
        printf("TIME SPENT ALLOCATING (TOTAL):  %3.10f ms \n", time);

    //ALLOC = false;
//  BEGIN COPYING DATA
    gettimeofday(&t1, 0);
    bool FORCE_COPY=true;

//  END step position
    if (db_flags.Fhkl_gradient_mode){
        for (int i=0; i < Npix_to_model; i++){
            cp.data_residual[i] = d_image.residual[i];
            cp.data_variance[i] = d_image.variance[i];
            cp.data_trusted[i] = d_image.trusted[i];
            cp.data_freq[i] = d_image.freq[i];
        }
    }

    if (db_flags.Fhkl_have_scale_factors && ALLOC){
        for (int i=0; i < db_cryst.FhklLinear_ASUid.size(); i++){
            cp.FhklLinear_ASUid[i] = db_cryst.FhklLinear_ASUid[i];
        }
    }

    if (db_flags.Fhkl_have_scale_factors){
        //SCITBX_ASSERT(db_beam.number_of_sources == db_beam.Fhkl_channels.size());
        for (int i=0; i < db_beam.number_of_sources; i++)
            cp.Fhkl_channels[i] = db_beam.Fhkl_channels[i];

        for (int i=0; i < d_image.Fhkl_scale.size(); i++){
            cp.Fhkl_scale[i] = d_image.Fhkl_scale[i];
            if (db_flags.Fhkl_gradient_mode){
                cp.Fhkl_scale_deriv[i] = 0;
            }
        }
    }

//  BEGIN sources
    if (db_cu_flags.update_sources || ALLOC || FORCE_COPY){
        for (int i=0; i< db_beam.number_of_sources; i++){
            VEC3 incident(db_beam.source_X[i], db_beam.source_Y[i], db_beam.source_Z[i]);
            incident /= incident.norm();
            cp.cu_source_X[i] = incident[0];
            cp.cu_source_Y[i] = incident[1];
            cp.cu_source_Z[i] = incident[2];
            cp.cu_source_I[i] = db_beam.source_I[i];
            cp.cu_source_lambda[i] = db_beam.source_lambda[i];
        }
        if(db_flags.verbose>1 )
          printf("H2D sources\n");
    }
//  END sources


//  UMATS
    if (db_cu_flags.update_umats || ALLOC||FORCE_COPY){
        for (int i=0; i< db_cryst.UMATS.size(); i++)
            cp.cu_UMATS[i] = db_cryst.UMATS[i];
        for (int i=0; i < db_cryst.UMATS_RXYZ.size(); i++)
            cp.cu_UMATS_RXYZ[i] = db_cryst.UMATS_RXYZ[i];
        for (int i=0; i < db_cryst.UMATS_RXYZ_prime.size(); i++)
            cp.cu_UMATS_RXYZ_prime[i] = db_cryst.UMATS_RXYZ_prime[i];
        for (int i=0; i < db_cryst.UMATS_RXYZ_dbl_prime.size(); i++)
            cp.cu_UMATS_RXYZ_dbl_prime[i] = db_cryst.UMATS_RXYZ_dbl_prime[i];
        if(db_flags.verbose>1)
            printf("H2D Done copying Umats\n") ;
    }
//  END UMATS


    if (db_cu_flags.update_umats || ALLOC||FORCE_COPY){
        MAT3 Amat_init = db_cryst.eig_U*db_cryst.eig_B*1e10*(db_cryst.eig_O.transpose());
        for(int i_mos =0; i_mos< db_cryst.UMATS_RXYZ.size(); i_mos++){
            cp.cu_AMATS[i_mos] = (db_cryst.UMATS_RXYZ[i_mos]*Amat_init).transpose();
            }
        if(db_flags.verbose>1)
            printf("H2D Done copying Amats\n") ;
    }


//  BMATS
    if(db_cu_flags.update_dB_mats || ALLOC || FORCE_COPY){
        for (int i=0; i< db_cryst.dB_Mats.size(); i++)
            cp.cu_dB_Mats[i] = db_cryst.dB_Mats[i];
        for (int i=0; i< db_cryst.dB2_Mats.size(); i++)
            cp.cu_dB2_Mats[i] = db_cryst.dB2_Mats[i];
        if(db_flags.verbose>1)
            printf("H2D Done copying dB_Mats\n") ;
    }
//  END BMATS


//  ROT MATS
    if(db_cu_flags.update_rotmats || ALLOC || FORCE_COPY){
        for (int i=0; i<db_cryst.RotMats.size(); i++)
            cp.cu_RotMats[i] = db_cryst.RotMats[i];
        for (int i=0; i<db_cryst.dRotMats.size(); i++)
            cp.cu_dRotMats[i] = db_cryst.dRotMats[i];
        for (int i=0; i<db_cryst.d2RotMats.size(); i++)
            cp.cu_d2RotMats[i] = db_cryst.d2RotMats[i];
        if (db_flags.verbose>1)
          printf("H2D Done copying rotmats\n");
    }
//  END ROT MATS

//  DETECTOR VECTORS
    if (db_cu_flags.update_detector || ALLOC || FORCE_COPY){
        for (int i=0; i<db_det.fdet_vectors.size(); i++){
            cp.cu_fdet_vectors[i] = db_det.fdet_vectors[i];
            cp.cu_sdet_vectors[i] = db_det.sdet_vectors[i];
            cp.cu_odet_vectors[i] = db_det.odet_vectors[i];
            cp.cu_pix0_vectors[i] = db_det.pix0_vectors[i];
        }
        for(int i=0; i < db_det.close_distances.size();i++){
            cp.cu_close_distances[i] = db_det.close_distances[i];
        }
        if (db_flags.verbose>1)
          printf("H2D Done copying detector vectors\n");
    }
//  END  DETECTOR VECTORS

    if ( ALLOC || FORCE_COPY){
      for(int i=0; i< db_cryst.nominal_hkl.size(); i++){
        cp.cu_nominal_hkl[i] = db_cryst.nominal_hkl[i];
      }
      for (int i=0; i< db_cryst.atom_data.size(); i++){
        cp.cu_atom_data[i] = db_cryst.atom_data[i];
      }
      if (db_flags.verbose>1)
        printf("H2D Done copying atom data\n");
      for(int i=0; i< db_cryst.fpfdp.size(); i++){
        cp.cu_fpfdp[i] = db_cryst.fpfdp[i];
      }
      for(int i=0; i< db_cryst.fpfdp_derivs.size(); i++){
        cp.cu_fpfdp_derivs[i] = db_cryst.fpfdp_derivs[i];
      }
      if (db_flags.verbose>1)
        printf("H2D Done copying fprime and fdblprime\n");
    }


//  BEGIN REFINEMENT FLAGS
    if (db_cu_flags.update_refine_flags || ALLOC || FORCE_COPY){
        for (int i=0; i<3; i++){
            cp.cu_refine_Umat[i] = db_flags.refine_Umat[i];
            cp.cu_refine_Ncells[i] = db_flags.refine_Ncells[i];
            cp.cu_refine_panel_origin[i] = db_flags.refine_panel_origin[i];
            cp.cu_refine_panel_rot[i] = db_flags.refine_panel_rot[i];
        }
        for(int i=0; i<2; i++)
            cp.cu_refine_lambda[i] = db_flags.refine_lambda[i];
        for(int i=0; i<6; i++)
            cp.cu_refine_Bmat[i] = db_flags.refine_Bmat[i];
        if (db_flags.verbose>1)
          printf("H2D Done copying refinement flags\n");
    }
//  END REFINEMENT FLAGS


//  BEGIN Fhkl
    if (db_cu_flags.update_Fhkl || ALLOC || FORCE_COPY){
        for(int i=0; i < db_cryst.FhklLinear.size(); i++){
          cp.cu_Fhkl[i] = db_cryst.FhklLinear[i];
          if (db_flags.complex_miller)
              cp.cu_Fhkl2[i] = db_cryst.Fhkl2Linear[i];
        }
        if (db_flags.verbose>1)
            printf("H2D Done copying step Fhkl\n");
    }
//  END Fhkl

//  BEGIN panel derivative vecs
    if(db_cu_flags.update_panel_deriv_vecs || ALLOC || FORCE_COPY){
        for (int i=0; i<db_det.dF_vecs.size(); i++){
            cp.cu_dF_vecs[i] = db_det.dF_vecs[i];
            cp.cu_dS_vecs[i] = db_det.dS_vecs[i];
        }
        if (db_flags.verbose>1)
            printf("H2D Done copying step panel derivative vectors\n");
    }
//  END panel derivative vecs

//  BEGIN panels fasts slows
    if (db_cu_flags.update_panels_fasts_slows || ALLOC || FORCE_COPY){
        for (int i=0; i< panels_fasts_slows.size(); i++)
            cp.cu_panels_fasts_slows[i] = panels_fasts_slows[i];
        if (db_flags.verbose>1)
            printf("H2D Done copying panels_fasts_slows\n");
    }
//  END panels fasts slows


    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if (TIMERS.recording) TIMERS.cuda_copy_to_dev += time;
    if(db_flags.verbose>1)
        printf("TIME SPENT COPYING DATA HOST->DEV:  %3.10f ms \n", time);

    cp.device_is_allocated = true;
    error_msg(hipGetLastError(), "after copy to device");

    gettimeofday(&t1, 0);

    int Npanels = db_det.fdet_vectors.size()/3;
    int num_atoms = db_cryst.atom_data.size()/5;
    if (db_cryst.fpfdp.size() == 0){ // note cannot use atom data if fpfdp is 0, make this cleaner
        num_atoms=0;
    }

    if (db_flags.use_diffuse) {
        if (db_cryst.laue_group_num < 1 || db_cryst.laue_group_num >14 ){
            throw std::string("Laue group number not in range 1-14");
        }
    }
    //int sm_size = number_of_sources*5*sizeof(CUDAREAL);
    //gpu_sum_over_steps<<<numblocks, blocksize, sm_size >>>(
    bool aniso_eta = db_cryst.UMATS_RXYZ.size() != db_cryst.UMATS_RXYZ_prime.size();
    bool use_nominal_hkl = !db_cryst.nominal_hkl.empty();
    gpu_sum_over_steps<<<numblocks, blocksize>>>(
        Npix_to_model, cp.cu_panels_fasts_slows,
        cp.cu_floatimage,
        cp.cu_wavelenimage,
        cp.cu_d_Umat_images, cp.cu_d2_Umat_images,
        cp.cu_d_Bmat_images, cp.cu_d2_Bmat_images,
        cp.cu_d_Ncells_images, cp.cu_d2_Ncells_images,
        cp.cu_d_fcell_images, cp.cu_d2_fcell_images,
        cp.cu_d_eta_images, cp.cu_d2_eta_images,
        cp.cu_d_lambda_images, cp.cu_d2_lambda_images,
        cp.cu_d_panel_rot_images, cp.cu_d2_panel_rot_images,
        cp.cu_d_panel_orig_images, cp.cu_d2_panel_orig_images,
        cp.cu_d_fp_fdp_images,
        db_steps.Nsteps, db_flags.printout_fpixel, db_flags.printout_spixel, db_flags.printout, db_cryst.default_F,
        db_det.oversample,  db_flags.oversample_omega, db_det.subpixel_size, db_det.pixel_size,
        db_det.detector_thickstep, db_det.detector_thick, cp.cu_close_distances, db_det.detector_attnlen,
        db_det.detector_thicksteps, db_beam.number_of_sources, db_cryst.phisteps, db_cryst.UMATS.size(),
        db_flags.use_lambda_coefficients, db_beam.lambda0, db_beam.lambda1,
        db_cryst.eig_U, db_cryst.eig_O, db_cryst.eig_B, db_cryst.RXYZ,
        cp.cu_dF_vecs,
        cp.cu_dS_vecs,
        cp.cu_UMATS_RXYZ,
        cp.cu_UMATS_RXYZ_prime,
        cp.cu_UMATS_RXYZ_dbl_prime,
        cp.cu_RotMats,
        cp.cu_dRotMats,
        cp.cu_d2RotMats,
        cp.cu_UMATS,
        cp.cu_dB_Mats,
        cp.cu_dB2_Mats,
        cp.cu_AMATS,
        cp.cu_source_X, cp.cu_source_Y, cp.cu_source_Z, cp.cu_source_lambda, cp.cu_source_I,
        db_beam.kahn_factor,
        db_cryst.Na, db_cryst.Nb, db_cryst.Nc,
        db_cryst.Nd, db_cryst.Ne, db_cryst.Nf,
        db_cryst.phi0, db_cryst.phistep,
        db_cryst.spindle_vec, db_beam.polarization_axis,
        db_cryst.h_range, db_cryst.k_range, db_cryst.l_range,
        db_cryst.h_max, db_cryst.h_min, db_cryst.k_max, db_cryst.k_min, db_cryst.l_max, db_cryst.l_min, db_cryst.dmin,
        db_cryst.fudge, db_flags.complex_miller, db_flags.verbose, db_flags.only_save_omega_kahn,
        db_flags.isotropic_ncells, db_flags.compute_curvatures,
        cp.cu_Fhkl, cp.cu_Fhkl2,
        cp.cu_refine_Bmat, cp.cu_refine_Ncells, db_flags.refine_Ncells_def, cp.cu_refine_panel_origin, cp.cu_refine_panel_rot,
        db_flags.refine_fcell, cp.cu_refine_lambda, db_flags.refine_eta, cp.cu_refine_Umat,
        cp.cu_fdet_vectors, cp.cu_sdet_vectors,
        cp.cu_odet_vectors, cp.cu_pix0_vectors,
        db_flags.nopolar, db_flags.point_pixel, db_beam.fluence, db_cryst.r_e_sqr, db_cryst.spot_scale, Npanels, aniso_eta, db_flags.no_Nabc_scale,
        cp.cu_fpfdp,  cp.cu_fpfdp_derivs, cp.cu_atom_data, num_atoms,
        db_flags.refine_fp_fdp, cp.cu_nominal_hkl, use_nominal_hkl,
        db_cryst.anisoU, db_cryst.anisoG, db_cryst.rotate_principal_axes,
        db_flags.use_diffuse, cp.cu_d_diffuse_gamma_images, cp.cu_d_diffuse_sigma_images,
        db_flags.refine_diffuse, db_flags.gamma_miller_units, db_flags.refine_Icell,
        db_flags.wavelength_img, db_cryst.laue_group_num, db_cryst.stencil_size,
        db_flags.Fhkl_gradient_mode, db_flags.Fhkl_errors_mode, db_flags.using_trusted_mask, db_beam.Fhkl_channels.empty(), db_flags.Fhkl_have_scale_factors,
        db_cryst.Num_ASU,
        cp.data_residual, cp.data_variance,
        cp.data_freq, cp.data_trusted,
        cp.FhklLinear_ASUid,
        cp.Fhkl_channels,
        cp.Fhkl_scale, cp.Fhkl_scale_deriv,
        db_cryst.xtal_shape==GAUSS_STAR,
        db_cryst.xtal_shape==SQUARE
        );

    error_msg(hipGetLastError(), "after kernel call");

    hipDeviceSynchronize();
    error_msg(hipGetLastError(), "after kernel completion");

    if(db_flags.verbose>1)
        printf("KERNEL_COMPLETE gpu_sum_over_steps\n");
    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if (TIMERS.recording) TIMERS.cuda_kernel += time;
    if(db_flags.verbose>1)
        printf("TIME SPENT(KERNEL):  %3.10f ms \n", time);

    gettimeofday(&t1, 0);
//  COPY BACK FROM DEVICE
    for (int i=0; i< Npix_to_model; i++){
        floatimage[i] = cp.cu_floatimage[i];
    }
    if(db_flags.wavelength_img){
        for (int i=0; i< 4*Npix_to_model; i++){
            d_image.wavelength[i] = cp.cu_wavelenimage[i];
        }
    }
    if (db_flags.refine_fcell){
        for (int i=0; i<Npix_to_model; i++){
            d_image.fcell[i] = cp.cu_d_fcell_images[i];
            d2_image.fcell[i] = cp.cu_d2_fcell_images[i];
        }
    }
    if (db_flags.Fhkl_gradient_mode){
        if (db_flags.Fhkl_errors_mode){
            for (int i=0; i < d_image.Fhkl_hessian.size(); i++)
                d_image.Fhkl_hessian[i]= cp.Fhkl_scale_deriv[i];
        }
        else{
            for (int i=0; i < d_image.Fhkl_scale_deriv.size(); i++)
                d_image.Fhkl_scale_deriv[i]= cp.Fhkl_scale_deriv[i];
        }
    }
    if (std::count(db_flags.refine_Umat.begin(), db_flags.refine_Umat.end(), true) > 0){
        for (int i=0; i<3*Npix_to_model; i++){
            d_image.Umat[i] = cp.cu_d_Umat_images[i];
            d2_image.Umat[i] = cp.cu_d2_Umat_images[i];
        }
    }
    if (std::count(db_flags.refine_panel_rot.begin(), db_flags.refine_panel_rot.end(), true) > 0){
        for (int i=0; i<3*Npix_to_model; i++)
            d_image.panel_rot[i] = cp.cu_d_panel_rot_images[i];
    }
    if (std::count(db_flags.refine_panel_origin.begin(), db_flags.refine_panel_origin.end(), true) > 0){
        for (int i=0; i<3*Npix_to_model; i++)
            d_image.panel_orig[i] = cp.cu_d_panel_orig_images[i];
    }
    if (db_flags.refine_eta){
        for (int i=0; i<3*Npix_to_model; i++){
            d_image.eta[i] = cp.cu_d_eta_images[i];
            d2_image.eta[i] = cp.cu_d2_eta_images[i];
        }
    }
    if (std::count(db_flags.refine_Ncells.begin(), db_flags.refine_Ncells.end(), true) > 0 || db_flags.refine_Ncells_def){
        for(int i=0; i<6*Npix_to_model; i++){
            d_image.Ncells[i] = cp.cu_d_Ncells_images[i];
            d2_image.Ncells[i] = cp.cu_d2_Ncells_images[i];
        }
    }
    if (db_flags.refine_diffuse){
        for(int i=0; i<3*Npix_to_model; i++){
            d_image.diffuse_gamma[i] = cp.cu_d_diffuse_gamma_images[i];
            d_image.diffuse_sigma[i] = cp.cu_d_diffuse_sigma_images[i];
        }
    }
    if (std::count(db_flags.refine_Bmat.begin(), db_flags.refine_Bmat.end(), true) > 0){
        for(int i=0; i<6*Npix_to_model; i++){
            d_image.Bmat[i] = cp.cu_d_Bmat_images[i];
            d2_image.Bmat[i] = cp.cu_d2_Bmat_images[i];
        }
    }
    if (std::count(db_flags.refine_lambda.begin(), db_flags.refine_lambda.end(), true) > 0){
        for(int i=0; i<2*Npix_to_model; i++)
            d_image.lambda[i] = cp.cu_d_lambda_images[i];
    }

    if (db_flags.refine_fp_fdp){
        for (int i=0; i< 2*Npix_to_model; i++)
            d_image.fp_fdp[i] = cp.cu_d_fp_fdp_images[i];
    }

    gettimeofday(&t2, 0);
    time = (1000000.0*(t2.tv_sec-t1.tv_sec) + t2.tv_usec-t1.tv_usec)/1000.0;
    if (TIMERS.recording) TIMERS.cuda_copy_from_dev += time;
    if(db_flags.verbose>1)
        printf("TIME SPENT COPYING BACK :  %3.10f ms \n", time);
    error_msg(hipGetLastError(), "After copy to host");


}


void freedom(diffBragg_cudaPointers& cp){

    if (cp.device_is_allocated){
        gpuErr(hipFree( cp.cu_floatimage));
        gpuErr(hipFree( cp.cu_wavelenimage));
        gpuErr(hipFree( cp.cu_d_Umat_images));
        gpuErr(hipFree( cp.cu_d_Bmat_images));
        gpuErr(hipFree( cp.cu_d_Ncells_images));
        gpuErr(hipFree( cp.cu_d_diffuse_gamma_images));
        gpuErr(hipFree( cp.cu_d_diffuse_sigma_images));
        gpuErr(hipFree( cp.cu_d2_Umat_images));
        gpuErr(hipFree( cp.cu_d2_Bmat_images));
        gpuErr(hipFree( cp.cu_d2_Ncells_images));
        gpuErr(hipFree( cp.cu_d_eta_images));
        gpuErr(hipFree( cp.cu_d2_eta_images));
        gpuErr(hipFree( cp.cu_d_fcell_images));
        gpuErr(hipFree( cp.cu_d2_fcell_images));
        gpuErr(hipFree( cp.cu_d_lambda_images));
        gpuErr(hipFree( cp.cu_d_panel_rot_images));
        gpuErr(hipFree( cp.cu_d_panel_orig_images));
        gpuErr(hipFree( cp.cu_d_sausage_XYZ_scale_images));
        gpuErr(hipFree( cp.cu_d_fp_fdp_images));

        gpuErr(hipFree(cp.cu_Fhkl));
        gpuErr(hipFree(cp.cu_Fhkl2));

        gpuErr(hipFree(cp.cu_fdet_vectors));
        gpuErr(hipFree(cp.cu_sdet_vectors));
        gpuErr(hipFree(cp.cu_odet_vectors));
        gpuErr(hipFree(cp.cu_pix0_vectors));
        gpuErr(hipFree(cp.cu_close_distances));
        gpuErr(hipFree(cp.cu_nominal_hkl));
        gpuErr(hipFree(cp.cu_atom_data));
        gpuErr(hipFree(cp.cu_fpfdp));
        gpuErr(hipFree(cp.cu_fpfdp_derivs));

        gpuErr(hipFree(cp.Fhkl_channels));
        gpuErr(hipFree(cp.cu_source_X));
        gpuErr(hipFree(cp.cu_source_Y));
        gpuErr(hipFree(cp.cu_source_Z));
        gpuErr(hipFree(cp.cu_source_I));
        gpuErr(hipFree(cp.cu_source_lambda));

        gpuErr(hipFree(cp.cu_UMATS));
        gpuErr(hipFree(cp.cu_UMATS_RXYZ));
        gpuErr(hipFree(cp.cu_AMATS));
        gpuErr(hipFree(cp.cu_UMATS_RXYZ_prime));
        gpuErr(hipFree(cp.cu_UMATS_RXYZ_dbl_prime));
        gpuErr(hipFree(cp.cu_RotMats));
        gpuErr(hipFree(cp.cu_dRotMats));
        gpuErr(hipFree(cp.cu_d2RotMats));
        gpuErr(hipFree(cp.cu_dB_Mats));
        gpuErr(hipFree(cp.cu_dB2_Mats));
        //gpuErr(hipFree(cp.cu_sausages_RXYZ));
        //gpuErr(hipFree(cp.cu_d_sausages_RXYZ));
        //gpuErr(hipFree(cp.cu_sausages_U));
        //gpuErr(hipFree(cp.cu_sausages_scale));

        gpuErr(hipFree(cp.cu_dF_vecs));
        gpuErr(hipFree(cp.cu_dS_vecs));

        gpuErr(hipFree(cp.cu_refine_Bmat));
        gpuErr(hipFree(cp.cu_refine_Umat));
        gpuErr(hipFree(cp.cu_refine_Ncells));
        gpuErr(hipFree(cp.cu_refine_lambda));
        gpuErr(hipFree(cp.cu_refine_panel_origin));
        gpuErr(hipFree(cp.cu_refine_panel_rot));

        gpuErr(hipFree(cp.cu_panels_fasts_slows));

        cp.device_is_allocated = false;
        cp.npix_allocated = 0;
    }

    if (cp.Fhkl_grad_arrays_allocated){
        gpuErr(hipFree(cp.data_trusted));
        gpuErr(hipFree(cp.data_freq));
        gpuErr(hipFree(cp.data_residual));
        gpuErr(hipFree(cp.data_variance));
        gpuErr(hipFree(cp.FhklLinear_ASUid));
        gpuErr(hipFree(cp.Fhkl_scale));
        gpuErr(hipFree(cp.Fhkl_scale_deriv));
        cp.Fhkl_grad_arrays_allocated=false;
    }
}
